#include "hip/hip_runtime.h"
/**
 * @file pso_optimizer.cu
 * @brief C++ Class Implementation for CUDA-accelerated PSO Spacecraft Attitude Control
 *
 * This implementation provides the complete class functionality for PSO optimization
 * of spacecraft attitude maneuvers with clean input/output interfaces.
 *
 * @author Leonardo Eitner
 * @date 11/09/2025
 * @version 2.0
 */

/*==============================================================================
 * INCLUDES
 *============================================================================*/
#include <toac/pso.h>

/*==============================================================================
 * CUDA CONSTANT MEMORY DECLARATIONS
 *============================================================================*/

/** @brief PSO algorithm parameters in device constant memory */
__constant__ real w_d, c1_d, c2_d;

/** @brief Physical constraint bounds in device constant memory */
__constant__ real max_torque_d, min_torque_d;
__constant__ real max_dt_d, min_dt_d;

/** @brief PSO velocity limits in device constant memory */
__constant__ real max_v_torque_d, max_v_dt_d;

/** @brief Problem dimensions in device constant memory */
__constant__ int particle_cnt_d, dimensions_d;

/** @brief Complete attitude parameters structure in device constant memory */
__constant__ attitude_params att_params_d;

/*==============================================================================
 * MATHEMATICAL UTILITY FUNCTIONS (CUDA KERNELS)
 *============================================================================*/

__host__ __device__ void skew_matrix_4(real *w, real *S)
{
    S[0] = REAL(0.0);
    S[1] = -w[0];
    S[2] = -w[1];
    S[3] = -w[2];
    S[4] = w[0];
    S[5] = REAL(0.0);
    S[6] = w[2];
    S[7] = -w[1];
    S[8] = w[1];
    S[9] = -w[2];
    S[10] = REAL(0.0);
    S[11] = w[0];
    S[12] = w[2];
    S[13] = w[1];
    S[14] = -w[0];
    S[15] = REAL(0.0);
}

__host__ __device__ void cross_product(real *a, real *b, real *result)
{
    result[0] = a[1] * b[2] - a[2] * b[1];
    result[1] = a[2] * b[0] - a[0] * b[2];
    result[2] = a[0] * b[1] - a[1] * b[0];
}

__host__ __device__ real quaternion_norm(real *q)
{
    return sqrt(q[0] * q[0] + q[1] * q[1] + q[2] * q[2] + q[3] * q[3]);
}

__host__ __device__ void attitude_dynamics(real *X, real *U, real *X_dot, attitude_params *params)
{
    real *q = X;
    real *w = &X[n_quat];

    // Quaternion kinematics: q̇ = 0.5 * S(ω) * q
    real S[16];
    skew_matrix_4(w, S);
    for (int i = 0; i < n_quat; i++)
    {
        X_dot[i] = REAL(0.5) * (S[i * 4] * q[0] + S[i * 4 + 1] * q[1] + S[i * 4 + 2] * q[2] + S[i * 4 + 3] * q[3]);
    }

    // Angular dynamics: ω̇ = I⁻¹ * (τ - ω × (I*ω))
    real Iw[n_vel] = {
        params->inertia[0] * w[0],
        params->inertia[1] * w[1],
        params->inertia[2] * w[2]};

    real w_cross_Iw[n_vel];
    cross_product(w, Iw, w_cross_Iw);

    for (int i = 0; i < n_vel; i++)
    {
        X_dot[n_quat + i] = (U[i] - w_cross_Iw[i]) / params->inertia[i];
    }
}

/*==============================================================================
 * NUMERICAL INTEGRATION SCHEMES
 *============================================================================*/

/**
 * @brief Fourth-order Runge-Kutta integration for attitude dynamics
 *
 * Provides high-accuracy numerical integration using the classical RK4 method:
 * k₁ = f(t, y)
 * k₂ = f(t + h/2, y + h*k₁/2)
 * k₃ = f(t + h/2, y + h*k₂/2)
 * k₄ = f(t + h, y + h*k₃)
 * y_{n+1} = y_n + h/6 * (k₁ + 2*k₂ + 2*k₃ + k₄)
 *
 * Used for final trajectory generation where accuracy is critical.
 * Automatically normalizes quaternion to maintain unit constraint.
 *
 * @param X Current state vector [q,ω] (7 elements)
 * @param U Control input vector [τx,τy,τz] (3 elements)
 * @param dt Integration time step (seconds)
 * @param X_next Output next state vector (7 elements)
 * @param params Spacecraft parameters
 */
__host__ __device__ void rk4(real *X, real *U, real dt, real *X_next, attitude_params *params)
{
    real k1[n_states], k2[n_states], k3[n_states], k4[n_states];
    real X_temp[n_states];

    // k1 = f(X, U)
    attitude_dynamics(X, U, k1, params);

    // k2 = f(X + dt/2*k1, U)
    for (int i = 0; i < n_states; i++)
        X_temp[i] = X[i] + dt / REAL(2.0) * k1[i];
    attitude_dynamics(X_temp, U, k2, params);

    // k3 = f(X + dt/2*k2, U)
    for (int i = 0; i < n_states; i++)
        X_temp[i] = X[i] + dt / REAL(2.0) * k2[i];
    attitude_dynamics(X_temp, U, k3, params);

    // k4 = f(X + dt*k3, U)
    for (int i = 0; i < n_states; i++)
        X_temp[i] = X[i] + dt * k3[i];
    attitude_dynamics(X_temp, U, k4, params);

    // Final integration step
    for (int i = 0; i < n_states; i++)
    {
        X_next[i] = X[i] + dt / REAL(6.0) * (k1[i] + REAL(2.0) * k2[i] + REAL(2.0) * k3[i] + k4[i]);
    }

    // Maintain quaternion unit constraint
    real q_norm = quaternion_norm(X_next);
    if (q_norm > REAL(1e-6))
    {
        for (int i = 0; i < n_quat; i++)
            X_next[i] /= q_norm;
    }
}

__host__ __device__ void euler(real *X, real *U, real dt, real *X_next, attitude_params *params)
{
    real X_dot[n_states];

    attitude_dynamics(X, U, X_dot, params);

    for (int i = 0; i < n_states; i++)
    {
        X_next[i] = X[i] + dt * X_dot[i];
    }

    real q_norm = quaternion_norm(X_next);
    if (q_norm > REAL(1e-6))
    {
        for (int i = 0; i < n_quat; i++)
            X_next[i] /= q_norm;
    }
}

__host__ __device__ real fit(real *solution_vector, int particle_id, attitude_params *params)
{
    real dt = solution_vector[PARTICLE_POS_IDX(particle_id, DT_IDX)];
    real total_time = dt * N_STEPS;

    // Decode initial control signs (map [-1,1] to {-1,+1})
    real initial_signs[n_controls];
    for (int axis = 0; axis < n_controls; axis++)
    {
        initial_signs[axis] = solution_vector[PARTICLE_POS_IDX(particle_id, SIGN_IDX(axis))];
        // real sign_val = solution_vector[PARTICLE_POS_IDX(particle_id, SIGN_IDX(axis))];
        // initial_signs[axis] = (sign_val >= REAL(0.0)) ? REAL(1.0) : REAL(-1.0);
    }

    // Decode and sort switch times for each axis (normalized to [0,1])
    real switch_times[n_controls][MAX_SWITCHES_PER_AXIS];
    int num_switches[n_controls];

    for (int axis = 0; axis < n_controls; axis++)
    {
        // Extract switch times for this axis
        real times[MAX_SWITCHES_PER_AXIS];
        for (int s = 0; s < MAX_SWITCHES_PER_AXIS; s++)
        {
            times[s] = solution_vector[PARTICLE_POS_IDX(particle_id, SWITCH_TIME_IDX(axis, s))];
            // Clamp to [0,1]
            times[s] = fmax(REAL(0.0), fmin(REAL(1.0), times[s]));
        }

        // Bubble sort (simple, efficient for small arrays)
        for (int i = 0; i < MAX_SWITCHES_PER_AXIS - 1; i++)
        {
            for (int j = 0; j < MAX_SWITCHES_PER_AXIS - i - 1; j++)
            {
                if (times[j] > times[j + 1])
                {
                    real temp = times[j];
                    times[j] = times[j + 1];
                    times[j + 1] = temp;
                }
            }
        }

        // Remove duplicates and count valid switches
        num_switches[axis] = 0;
        for (int s = 0; s < MAX_SWITCHES_PER_AXIS; s++)
        {
            // Only count if different from previous and not at boundaries
            if (times[s] > REAL(0.01) && times[s] < REAL(0.99))
            {
                if (num_switches[axis] == 0 ||
                    fabs(times[s] - switch_times[axis][num_switches[axis] - 1]) > REAL(0.01))
                {
                    switch_times[axis][num_switches[axis]] = times[s];
                    num_switches[axis]++;
                }
            }
        }
    }

    // Simulate trajectory with bang-bang control
    real constraints_violation = REAL(0.0);
    real X[n_states], X_next[n_states];

    for (int i = 0; i < n_quat; i++)
        X[i] = params->initial_quat[i];
    for (int i = 0; i < n_vel; i++)
        X[n_quat + i] = params->initial_omega[i];

    real current_time = REAL(0.0);

    for (int step = 0; step < N_STEPS; step++)
    {
        real step_start_time = current_time / total_time; // Normalized time [0,1]
        real U[n_controls];

        // Determine control for each axis based on switch times
        for (int axis = 0; axis < n_controls; axis++)
        {
            real control_sign = initial_signs[axis];

            // Count how many switches have occurred by this time
            for (int s = 0; s < num_switches[axis]; s++)
            {
                if (step_start_time >= switch_times[axis][s])
                {
                    control_sign *= REAL(-1.0); // Flip sign at each switch
                }
            }

            // Apply bang-bang control at max torque
            U[axis] = control_sign * params->max_torque;
        }

        rk4(X, U, dt, X_next, params);

        real q_norm = quaternion_norm(X_next);
        constraints_violation -= QUAT_NORM_PENALTY * fabs(q_norm - REAL(1.0));

        for (int i = 0; i < n_states; i++)
            X[i] = X_next[i];
        current_time += dt;
    }

    // Final state error
    real final_error = REAL(0.0);
    real diff;
    for (int i = 0; i < n_quat; i++)
    {
        diff = X[i] - params->target_quat[i];
        final_error += diff * diff;
    }
    for (int i = 0; i < n_vel; i++)
    {
        diff = X[n_quat + i] - params->target_omega[i];
        final_error += diff * diff;
    }
    final_error = sqrt(final_error);

    constraints_violation -= FINAL_STATE_PENALTY * final_error;
    constraints_violation -= DT_PENALTY * total_time;

    return constraints_violation;
}

/*==============================================================================
 * CUDA KERNEL IMPLEMENTATIONS
 *============================================================================*/

__global__ void move(real *position_d, real *velocity_d, real *fitness_d,
                     real *pbest_pos_d, real *pbest_fit_d,
                     particle_gbest *gbest_d, real *aux, real *aux_pos)
{

    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    int tidx = threadIdx.x;

    extern __shared__ real sharedMemory[];
    real *privateBestQueue = (real *)sharedMemory;
    int *privateBestParticleQueue = (int *)&sharedMemory[blockDim.x];
    __shared__ unsigned int queue_num;

    if (particle_id >= particle_cnt_d)
        return;

    if (tidx == 0)
        queue_num = 0;
    __syncthreads();

    hiprandState state1, state2;
    hiprand_init((unsigned long long)clock() + particle_id * 2, 0, 0, &state1);
    hiprand_init((unsigned long long)clock() + particle_id * 2 + 1, 0, 0, &state2);

    real w = w_d, c1 = c1_d, c2 = c2_d;
    if (DEC_INERTIA)
    {
        w = w_d - (w_d - MIN_W) * (real)particle_id / N_PARTICLES;
    }

    if (DEC_C1)
    {
        c1 = c1_d - (c1_d - MIN_C1) * (real)particle_id / N_PARTICLES;
    }

    if (DEC_C2)
    {
        c2 = c2_d - (c2_d - MIN_C2) * (real)particle_id / N_PARTICLES;
    }

    for (int dim = 0; dim < dimensions_d; dim++)
    {
        int pos_idx = PARTICLE_POS_IDX(particle_id, dim);

        real pos = position_d[pos_idx];
        real vel = velocity_d[pos_idx];
        real pbest_pos = pbest_pos_d[pos_idx];
        real gbest_pos = gbest_d->position[dim];

        vel = w * vel +
              c1 * CURAND(state1) * (pbest_pos - pos) +
              c2 * CURAND(state2) * (gbest_pos - pos);

        // Apply bounds based on dimension type
        if (dim < N_SIGNS)
        {   
            // Clamp velocity for binary dimensions
            vel = fmax(REAL(-6.0), fmin(REAL(6.0), vel));
            
            // Sigmoid transfer function: S(v) = 1/(1 + exp(-alpha*v))
            real sigmoid = REAL(1.0) / (REAL(1.0) + exp(-SIGMOID_ALPHA * vel));
            
            // Probabilistic position update
            if (CURAND(state1) < sigmoid) {
                pos = REAL(1.0);   // Positive control
            } else {
                pos = REAL(-1.0);  // Negative control
            }
            // Sign dimensions: keep in [-1, 1]
            // vel = fmax(REAL(-2.0), fmin(REAL(2.0), vel));
            // pos = pos + vel;
            // pos = fmax(REAL(-1.0), fmin(REAL(1.0), pos));
        }
        else if (dim < N_SIGNS + N_SWITCH_TIMES)
        {
            // Switch time dimensions: keep in [0, 1]
            vel = fmax(REAL(-1.0), fmin(REAL(1.0), vel));
            pos = pos + vel;
            pos = fmax(REAL(0.0), fmin(REAL(1.0), pos));
        }
        else
        {
            // dt dimension: use existing bounds
            vel = fmax(-max_v_dt_d, fmin(max_v_dt_d, vel));
            pos = pos + vel;
            pos = fmax(min_dt_d, fmin(max_dt_d, pos));
        }

        position_d[pos_idx] = pos;
        velocity_d[pos_idx] = vel;
    }

    real new_fitness = fit(position_d, particle_id, &att_params_d);
    fitness_d[particle_id] = new_fitness;

    if (new_fitness > pbest_fit_d[particle_id])
    {
        pbest_fit_d[particle_id] = new_fitness;
        for (int dim = 0; dim < dimensions_d; dim++)
        {
            pbest_pos_d[PARTICLE_POS_IDX(particle_id, dim)] =
                position_d[PARTICLE_POS_IDX(particle_id, dim)];
        }
    }

    __syncthreads();

    if (new_fitness > gbest_d->fitness)
    {
        unsigned int my_index = atomicAdd(&queue_num, 1);
        if (my_index < blockDim.x)
        {
            privateBestQueue[my_index] = new_fitness;
            privateBestParticleQueue[my_index] = particle_id;
        }
    }

    __syncthreads();

    if (tidx == 0)
    {
        aux[blockIdx.x] = -REAL_MAX;
        aux_pos[blockIdx.x] = -1;

        if (queue_num > 0)
        {
            real best_fitness = privateBestQueue[0];
            int best_idx = 0;

            for (unsigned int i = 1; i < queue_num && i < blockDim.x; i++)
            {
                if (privateBestQueue[i] > best_fitness)
                {
                    best_fitness = privateBestQueue[i];
                    best_idx = i;
                }
            }
            aux[blockIdx.x] = best_fitness;
            aux_pos[blockIdx.x] = privateBestParticleQueue[best_idx];
        }
    }
}

__global__ void findBest(particle_gbest *gbest, real *aux, real *aux_pos, real *position_d)
{
    int tid = threadIdx.x;

    real my_fitness = (tid < BlocksPerGrid) ? aux[tid] : -REAL_MAX;
    int my_particle = (tid < BlocksPerGrid) ? (int)aux_pos[tid] : -1;

    for (int offset = 16; offset > 0; offset /= 2)
    {
        real other_fitness = __shfl_down_sync(0xffffffff, my_fitness, offset);
        int other_particle = __shfl_down_sync(0xffffffff, my_particle, offset);
        if (other_fitness > my_fitness)
        {
            my_fitness = other_fitness;
            my_particle = other_particle;
        }
    }

    if (tid == 0 && my_fitness > gbest->fitness)
    {
        gbest->fitness = my_fitness;
        if (my_particle >= 0)
        {
            for (int dim = 0; dim < DIMENSIONS; dim++)
            {
                gbest->position[dim] = position_d[PARTICLE_POS_IDX(my_particle, dim)];
            }
        }
        __threadfence();
    }
}

/*==============================================================================
 * PSO OPTIMIZER CLASS IMPLEMENTATION
 *============================================================================*/
PSOOptimizer::PSOOptimizer(casadi::DM &state_matrix, casadi::DM &input_matrix, casadi::DM &dt_matrix, bool verbose)
    : configured_(false), results_valid_(false), max_iterations_(ITERATIONS), num_particles_(N_PARTICLES), inertia_weight_(W), cognitive_weight_(C1), social_weight_(C2), particles_(nullptr), position_d_(nullptr), velocity_d_(nullptr), fitness_d_(nullptr), pbest_pos_d_(nullptr), pbest_fit_d_(nullptr), gbest_d_(nullptr), aux_(nullptr), aux_pos_(nullptr), lhs_samples_(nullptr), lhs_generated_(false), verbose_(verbose), X(state_matrix), U(input_matrix), dt(dt_matrix)
{
    // Initialize CUDA events
    if (!handleCudaError(hipEventCreate(&start_event_), __FILE__, __LINE__) ||
        !handleCudaError(hipEventCreate(&stop_event_), __FILE__, __LINE__))
    {
        std::cerr << "Failed to create CUDA events" << std::endl;
    }

    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);

    if ((X.size1() != n_states || X.size2() != N_STEPS + 1) ||
        (U.size1() != n_controls || U.size2() != N_STEPS) ||
        (dt.size1() != N_STEPS || dt.size2() != 1))
    {
        std::cerr << "Error: Output matrices have incorrect dimensions." << std::endl;
        std::cerr << "Expected dimensions | Given dimensions: "
                  << "X: (" << n_states << ", " << N_STEPS + 1 << ") | (" << X.size1() << ", " << X.size2() << "), "
                  << "U: (" << n_controls << ", " << N_STEPS << ") | (" << U.size1() << ", " << U.size2() << "), "
                  << "dt: (" << N_STEPS << ", 1) | (" << dt.size1() << ", " << dt.size2() << ")" << std::endl;
        cleanup();
        return;
    }

    // Initialize attitude parameters with default spacecraft values
    memset(&att_params_, 0, sizeof(attitude_params));

    att_params_.inertia[0] = static_cast<real>(i_x);
    att_params_.inertia[1] = static_cast<real>(i_y);
    att_params_.inertia[2] = static_cast<real>(i_z);
    att_params_.max_torque = static_cast<real>(tau_max);
    att_params_.min_torque = -static_cast<real>(tau_max);
    att_params_.max_dt = static_cast<real>(dt_max);
    att_params_.min_dt = static_cast<real>(dt_min);

    // Initialize velocity limits
    max_v_torque_ = REAL(2.0) * att_params_.max_torque;
    max_v_dt_ = att_params_.max_dt - att_params_.min_dt;

    // Allocate LHS samples storage
    lhs_samples_ = new real *[num_particles_];
    for (int i = 0; i < num_particles_; i++)
    {
        lhs_samples_[i] = new real[DIMENSIONS];
    }

    // Allocate host particle structure (once)
    particles_ = (particle *)malloc(sizeof(particle));
    if (!particles_)
    {
        std::cerr << "Failed to allocate particle structure" << std::endl;
        cleanup();
        return;
    }

    particles_->position = (real *)malloc(sizeof(real) * num_particles_ * DIMENSIONS);
    particles_->velocity = (real *)malloc(sizeof(real) * num_particles_ * DIMENSIONS);
    particles_->pbest_pos = (real *)malloc(sizeof(real) * num_particles_ * DIMENSIONS);
    particles_->fitness = (real *)malloc(sizeof(real) * num_particles_);
    particles_->pbest_fit = (real *)malloc(sizeof(real) * num_particles_);

    if (!particles_->position || !particles_->velocity || !particles_->pbest_pos ||
        !particles_->fitness || !particles_->pbest_fit)
    {
        std::cerr << "Failed to allocate particle arrays" << std::endl;
        cleanup();
        return;
    }

    // One-time GPU initialization
    if (!allocateDeviceMemory() || !copyImmutableConstants())
    {
        std::cerr << "GPU initialization failed" << std::endl;
        cleanup();
        return;
    }

    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&setup_time_, start_event_, stop_event_), __FILE__, __LINE__);
    setup_time_ /= 1000.0f;
}

PSOOptimizer::~PSOOptimizer()
{
    cleanup();
}

bool PSOOptimizer::allocateDeviceMemory()
{
    size_t particle_data_size = sizeof(real) * num_particles_ * DIMENSIONS;

    if (!handleCudaError(hipMalloc((void **)&position_d_, particle_data_size), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void **)&velocity_d_, particle_data_size), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void **)&pbest_pos_d_, particle_data_size), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void **)&fitness_d_, sizeof(real) * num_particles_), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void **)&pbest_fit_d_, sizeof(real) * num_particles_), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void **)&gbest_d_, sizeof(particle_gbest)), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void **)&aux_, sizeof(real) * BlocksPerGrid), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void **)&aux_pos_, sizeof(real) * BlocksPerGrid), __FILE__, __LINE__))
    {
        return false;
    }

    return true;
}

void PSOOptimizer::setPSOParameters(int max_iterations,
                                    double inertia_weight, double cognitive_weight, double social_weight)
{

    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);

    max_iterations_ = max_iterations;
    inertia_weight_ = static_cast<real>(inertia_weight);
    cognitive_weight_ = static_cast<real>(cognitive_weight);
    social_weight_ = static_cast<real>(social_weight);

    copyImmutableConstants();

    results_valid_ = false;
    // Stop timing
    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);

    // Calculate execution time
    float temp_time;
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&temp_time, start_event_, stop_event_), __FILE__, __LINE__);
    setup_time_ += temp_time; // Accumulate
}

bool PSOOptimizer::copyImmutableConstants()
{
    // Copy PSO parameters
    if (!handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(w_d), &inertia_weight_, sizeof(real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(c1_d), &cognitive_weight_, sizeof(real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(c2_d), &social_weight_, sizeof(real)), __FILE__, __LINE__))
    {
        return false;
    }

    // Copy physical constraints (immutable)
    if (!handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(max_torque_d), &att_params_.max_torque, sizeof(real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(min_torque_d), &att_params_.min_torque, sizeof(real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(max_dt_d), &att_params_.max_dt, sizeof(real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(min_dt_d), &att_params_.min_dt, sizeof(real)), __FILE__, __LINE__))
    {
        return false;
    }

    // Copy velocity limits
    if (!handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(max_v_torque_d), &max_v_torque_, sizeof(real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(max_v_dt_d), &max_v_dt_, sizeof(real)), __FILE__, __LINE__))
    {
        return false;
    }

    // Copy dimensions
    if (!handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(particle_cnt_d), &num_particles_, sizeof(int)), __FILE__, __LINE__))
    {
        return false;
    }

    int dimensions = DIMENSIONS;
    if (!handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(dimensions_d), &dimensions, sizeof(int)), __FILE__, __LINE__))
    {
        return false;
    }

    return true;
}

bool PSOOptimizer::copyMutableStateParameters()
{
    // Only copy the att_params structure (contains initial/target states)
    return handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(att_params_d), &att_params_, sizeof(attitude_params)), __FILE__, __LINE__);
}

void PSOOptimizer::setStates(const double *initial_state, const double *target_state)
{
    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);
    // Update initial and target states
    for (int i = 0; i < n_quat; i++)
    {
        att_params_.initial_quat[i] = static_cast<real>(initial_state[i]);
        att_params_.target_quat[i] = static_cast<real>(target_state[i]);
    }
    for (int i = 0; i < n_vel; i++)
    {
        att_params_.initial_omega[i] = static_cast<real>(initial_state[i + n_quat]);
        att_params_.target_omega[i] = static_cast<real>(target_state[i + n_quat]);
    }

    // Copy updated parameters to device constant memory
    copyMutableStateParameters();

    results_valid_ = false;

    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);

    // Calculate execution time
    float temp_time;
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&temp_time, start_event_, stop_event_), __FILE__, __LINE__);
    setup_time_ += temp_time; // Accumulate
}

bool PSOOptimizer::initializeParticles(bool regenerate_lhs)
{
    srand((unsigned)time(NULL));

    // Generate or reuse LHS samples
    if (regenerate_lhs || !lhs_generated_)
    {
        generateLHSSamples(lhs_samples_);
        lhs_generated_ = true;
    }

    // Initialize global best
    gbest_.fitness = -REAL_MAX;

    // Initialize each particle using LHS samples
    int best_particle = 0;
    for (int i = 0; i < num_particles_; i++)
    {
        // Initialize sign variables [-1, 1]
        for (int dim = 0; dim < N_SIGNS; dim++)
        {
            int idx = PARTICLE_POS_IDX(i, dim);
            // particles_->position[idx] = lhs_samples_[i][dim] * REAL(2.0) - REAL(1.0); // Map [0,1] to [-1,1]
            particles_->position[idx] = (lhs_samples_[i][dim] > REAL(0.5)) ? REAL(1.0) : REAL(-1.0);
            particles_->velocity[idx] = (lhs_samples_[i][dim] - REAL(0.5)) * REAL(4.0);
            particles_->pbest_pos[idx] = particles_->position[idx];
        }

        // Initialize switch time variables [0, 1]
        for (int dim = N_SIGNS; dim < N_SIGNS + N_SWITCH_TIMES; dim++)
        {
            int idx = PARTICLE_POS_IDX(i, dim);
            particles_->position[idx] = lhs_samples_[i][dim]; // Already in [0,1]
            particles_->velocity[idx] = (lhs_samples_[i][dim] - REAL(0.5)) * REAL(2.0);
            particles_->pbest_pos[idx] = particles_->position[idx];
        }

        // Initialize dt (unchanged)
        int dt_idx = PARTICLE_POS_IDX(i, DT_IDX);
        real dt_range = att_params_.max_dt - att_params_.min_dt;
        particles_->position[dt_idx] = lhs_samples_[i][DT_IDX] * dt_range + att_params_.min_dt;
        particles_->velocity[dt_idx] = (lhs_samples_[i][DT_IDX] - REAL(0.5)) * REAL(2.0) * max_v_dt_;
        particles_->pbest_pos[dt_idx] = particles_->position[dt_idx];

        // Evaluate initial fitness
        particles_->fitness[i] = fit(particles_->position, i, &att_params_);
        particles_->pbest_fit[i] = particles_->fitness[i];

        // Track global best
        if (i == 0 || particles_->pbest_fit[i] > gbest_.fitness)
        {
            best_particle = i;
            gbest_.fitness = particles_->fitness[i];
        }
    }

    // Copy global best position
    for (int dim = 0; dim < DIMENSIONS; dim++)
    {
        int idx = PARTICLE_POS_IDX(best_particle, dim);
        gbest_.position[dim] = particles_->position[idx];
    }

    return true;
}

/**
 * Generate Latin Hypercube Samples for particle initialization
 * @param samples Output matrix [num_particles_][DIMENSIONS]
 */
void PSOOptimizer::generateLHSSamples(real **samples)
{
    // Generate LHS samples in [0,1] for each dimension
    for (int dim = 0; dim < DIMENSIONS; dim++)
    {
        std::vector<real> intervals(num_particles_);

        // Create stratified intervals
        for (int i = 0; i < num_particles_; i++)
        {
            real interval_start = static_cast<real>(i) / num_particles_;
            real interval_width = REAL(1.0) / num_particles_;
            intervals[i] = interval_start + RND() * interval_width;
        }

        // Shuffle to break correlations
        for (int i = num_particles_ - 1; i > 0; i--)
        {
            int j = rand() % (i + 1);
            std::swap(intervals[i], intervals[j]);
        }

        // Assign to samples matrix
        for (int i = 0; i < num_particles_; i++)
        {
            samples[i][dim] = intervals[i];
        }
    }
}

bool PSOOptimizer::optimize(bool regenerate_lhs)
{

    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);

    // Initialize particles (regenerate LHS based on parameter)
    if (!initializeParticles(regenerate_lhs))
    {
        std::cerr << "Particle initialization failed" << std::endl;
        return false;
    }

    // Copy initial data to device
    size_t particle_data_size = sizeof(real) * num_particles_ * DIMENSIONS;
    if (!handleCudaError(hipMemcpy(position_d_, particles_->position, particle_data_size, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(velocity_d_, particles_->velocity, particle_data_size, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(pbest_pos_d_, particles_->pbest_pos, particle_data_size, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(fitness_d_, particles_->fitness, sizeof(real) * num_particles_, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(pbest_fit_d_, particles_->pbest_fit, sizeof(real) * num_particles_, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(gbest_d_, &gbest_, sizeof(particle_gbest), hipMemcpyHostToDevice), __FILE__, __LINE__))
    {
        return false;
    }

    if (verbose_)
    {
        std::cout << "Starting PSO optimization..." << std::endl;
        std::cout << "LHS: " << (regenerate_lhs ? "Regenerated" : "Reused") << std::endl;
        std::cout << "Initial best fitness: " << gbest_.fitness << std::endl;
    }

    // Main optimization loop
    int shared_mem_size = sizeof(real) * ThreadsPerBlock + sizeof(int) * ThreadsPerBlock;

    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);
    float temp_time;
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&temp_time, start_event_, stop_event_), __FILE__, __LINE__);
    setup_time_ += temp_time / 1000.0f; // Convert ms to seconds
    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);

    for (int iter = 0; iter < max_iterations_; iter++)
    {
        move<<<BlocksPerGrid, ThreadsPerBlock, shared_mem_size>>>(
            position_d_, velocity_d_, fitness_d_, pbest_pos_d_, pbest_fit_d_,
            gbest_d_, aux_, aux_pos_);

        if (!handleCudaError(hipDeviceSynchronize(), __FILE__, __LINE__))
        {
            return false;
        }

        findBest<<<1, 32>>>(gbest_d_, aux_, aux_pos_, position_d_);

        if (!handleCudaError(hipDeviceSynchronize(), __FILE__, __LINE__))
        {
            return false;
        }

        if (verbose_ && (iter % 100 == 0 || iter == max_iterations_ - 1))
        {
            particle_gbest current_best;
            if (handleCudaError(hipMemcpy(&current_best, gbest_d_, sizeof(particle_gbest), hipMemcpyDeviceToHost), __FILE__, __LINE__))
            {
                std::cout << "Iteration " << iter << ": Best fitness = " << current_best.fitness << std::endl;
            }
        }
    }

    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&exec_time_, start_event_, stop_event_), __FILE__, __LINE__);

    // Copy final results
    if (!handleCudaError(hipMemcpy(&gbest_, gbest_d_, sizeof(particle_gbest), hipMemcpyDeviceToHost), __FILE__, __LINE__))
    {
        std::cerr << "Error copying final results" << std::endl;
        return false;
    }

    dt_opt_ = gbest_.position[DT_IDX];
    total_time_ = dt_opt_ * N_STEPS;
    final_fitness_ = gbest_.fitness;
    exec_time_ = exec_time_ / 1000.0f;

    if (!extractResults())
    {
        std::cerr << "Error extracting results or invalid results" << std::endl;
        return false;
    }

    results_valid_ = true;
    configured_ = true;

    if (verbose_)
    {
        printResults();
    }

    return true;
}

bool PSOOptimizer::extractResults()
{
    double dt_double = static_cast<double>(dt_opt_);

    // Decode initial signs
    real initial_signs[n_controls];
    for (int axis = 0; axis < n_controls; axis++)
    {
        real sign_val = gbest_.position[SIGN_IDX(axis)];
        initial_signs[axis] = (sign_val >= REAL(0.0)) ? REAL(1.0) : REAL(-1.0);
    }

    // Decode and sort switch times
    real switch_times[n_controls][MAX_SWITCHES_PER_AXIS];
    int num_switches[n_controls];

    for (int axis = 0; axis < n_controls; axis++)
    {
        real times[MAX_SWITCHES_PER_AXIS];
        for (int s = 0; s < MAX_SWITCHES_PER_AXIS; s++)
        {
            times[s] = gbest_.position[SWITCH_TIME_IDX(axis, s)];
            times[s] = std::max(REAL(0.0), std::min(REAL(1.0), times[s]));
        }

        std::sort(times, times + MAX_SWITCHES_PER_AXIS);

        num_switches[axis] = 0;
        for (int s = 0; s < MAX_SWITCHES_PER_AXIS; s++)
        {
            if (times[s] > REAL(0.01) && times[s] < REAL(0.99))
            {
                if (num_switches[axis] == 0 ||
                    std::abs(times[s] - switch_times[axis][num_switches[axis] - 1]) > REAL(0.01))
                {
                    switch_times[axis][num_switches[axis]] = times[s];
                    num_switches[axis]++;
                }
            }
        }
    }

    // Simulate trajectory
    real current_state[n_states], next_state[n_states];
    real total_time = dt_opt_ * N_STEPS;

    for (int i = 0; i < n_quat; i++)
    {
        current_state[i] = att_params_.initial_quat[i];
        X(i, 0) = static_cast<double>(current_state[i]);
    }
    for (int i = 0; i < n_vel; i++)
    {
        current_state[n_quat + i] = att_params_.initial_omega[i];
        X(n_quat + i, 0) = static_cast<double>(current_state[n_quat + i]);
    }

    real current_time = REAL(0.0);

    for (int step = 0; step < N_STEPS; step++)
    {
        real step_start_time = current_time / total_time;
        real controls[n_controls];

        for (int axis = 0; axis < n_controls; axis++)
        {
            real control_sign = initial_signs[axis];

            for (int s = 0; s < num_switches[axis]; s++)
            {
                if (step_start_time >= switch_times[axis][s])
                {
                    control_sign *= REAL(-1.0); // Flip sign at each switch
                }
            }

            controls[axis] = control_sign * att_params_.max_torque;
            U(axis, step) = static_cast<double>(controls[axis]);
        }

        rk4(current_state, controls, dt_opt_, next_state, &att_params_);

        for (int i = 0; i < n_states; i++)
        {
            current_state[i] = next_state[i];
            X(i, step + 1) = static_cast<double>(current_state[i]);
        }

        dt(step) = dt_double;
        current_time += dt_opt_;
    }

    // // Validate final state
    // real final_error = REAL(0.0);
    // real diff;
    // for (int i = 0; i < n_quat; i++)
    // {
    //     diff = current_state[i] - att_params_.target_quat[i];
    //     final_error += diff * diff;
    // }
    // for (int i = 0; i < n_vel; i++)
    // {
    //     diff = current_state[n_quat + i] - att_params_.target_omega[i];
    //     final_error += diff * diff;
    // }
    // final_error = sqrt(final_error);

    // if (final_error > REAL(1e-3))
    // {
    //     std::cerr << "Warning: Final state error: " << final_error << std::endl;
    //     return false;
    // }
    return true;
}

bool PSOOptimizer::getStats(double &final_fitness, double &setup_time, double &exec_time) const
{
    if (!results_valid_)
    {
        std::cerr << "Warning: No valid results available. Call optimize() first." << std::endl;
        return false;
    }

    final_fitness = static_cast<double>(final_fitness_);
    setup_time = static_cast<double>(setup_time_);
    exec_time = static_cast<double>(exec_time_);

    return true;
}

void PSOOptimizer::printResults() const
{
    if (!results_valid_)
    {
        std::cout << "No valid results available." << std::endl;
        return;
    }

    std::cout << "\n=== PSO Optimization Results ===" << std::endl;
    std::cout << "Final fitness: " << std::setprecision(6) << final_fitness_ << std::endl;
    std::cout << "Total maneuver time: " << total_time_ << " seconds" << std::endl;
    std::cout << "Time step: " << dt_opt_ << " seconds" << std::endl;
    std::cout << "Execution time: " << exec_time_ << " seconds" << std::endl;
    std::cout << "Setup time: " << setup_time_ << " seconds" << std::endl;
    std::cout << "Total computation time: " << (setup_time_ + exec_time_) << " seconds" << std::endl;
    std::cout << "===============================\n"
              << std::endl;
}

void PSOOptimizer::reset()
{
    cleanup();
    results_valid_ = false;
    configured_ = false;
}

void PSOOptimizer::cleanup()
{
    // Free LHS samples
    if (lhs_samples_)
    {
        for (int i = 0; i < num_particles_; i++)
        {
            if (lhs_samples_[i])
                delete[] lhs_samples_[i];
        }
        delete[] lhs_samples_;
        lhs_samples_ = nullptr;
    }

    // Free host memory
    if (particles_)
    {
        if (particles_->position)
            free(particles_->position);
        if (particles_->velocity)
            free(particles_->velocity);
        if (particles_->pbest_pos)
            free(particles_->pbest_pos);
        if (particles_->fitness)
            free(particles_->fitness);
        if (particles_->pbest_fit)
            free(particles_->pbest_fit);
        free(particles_);
        particles_ = nullptr;
    }

    // Free device memory
    if (position_d_)
    {
        hipFree(position_d_);
        position_d_ = nullptr;
    }
    if (velocity_d_)
    {
        hipFree(velocity_d_);
        velocity_d_ = nullptr;
    }
    if (fitness_d_)
    {
        hipFree(fitness_d_);
        fitness_d_ = nullptr;
    }
    if (pbest_pos_d_)
    {
        hipFree(pbest_pos_d_);
        pbest_pos_d_ = nullptr;
    }
    if (pbest_fit_d_)
    {
        hipFree(pbest_fit_d_);
        pbest_fit_d_ = nullptr;
    }
    if (gbest_d_)
    {
        hipFree(gbest_d_);
        gbest_d_ = nullptr;
    }
    if (aux_)
    {
        hipFree(aux_);
        aux_ = nullptr;
    }
    if (aux_pos_)
    {
        hipFree(aux_pos_);
        aux_pos_ = nullptr;
    }

    if (start_event_)
    {
        hipEventDestroy(start_event_);
        start_event_ = nullptr;
    }
    if (stop_event_)
    {
        hipEventDestroy(stop_event_);
        stop_event_ = nullptr;
    }
}

bool PSOOptimizer::handleCudaError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Error: " << hipGetErrorString(err)
                  << " in " << file << " at line " << line << std::endl;
        return false;
    }
    return true;
}