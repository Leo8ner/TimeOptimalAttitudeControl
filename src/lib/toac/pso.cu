#include "hip/hip_runtime.h"
/**
 * @file pso_optimizer.cu
 * @brief C++ Class Implementation for CUDA-accelerated PSO Spacecraft Attitude Control
 * 
 * This implementation provides the complete class functionality for PSO optimization
 * of spacecraft attitude maneuvers with clean input/output interfaces.
 * 
 * @author Leonardo Eitner
 * @date 11/09/2025
 * @version 2.0
 */

/*==============================================================================
 * INCLUDES
 *============================================================================*/
#include <toac/pso.h>
 
/*==============================================================================
 * CUDA CONSTANT MEMORY DECLARATIONS
 *============================================================================*/

/** @brief PSO algorithm parameters in device constant memory */
__constant__ my_real w_d, c1_d, c2_d;
__constant__ my_real min_w_d, min_c1_d, min_c2_d;
__constant__ bool decay_w_d, decay_c1_d, decay_c2_d;
__constant__ my_real alpha_d, saturation_d;

/** @brief Physical constraint bounds in device constant memory */
__constant__ my_real max_torque_d, min_torque_d;
__constant__ my_real max_dt_d, min_dt_d;

/** @brief PSO velocity limits in device constant memory */
__constant__ my_real max_v_torque_d, max_v_dt_d;

/** @brief Problem dimensions in device constant memory */
__constant__ int particle_cnt_d, dimensions_d;

/** @brief Complete attitude parameters structure in device constant memory */
__constant__ attitude_params att_params_d;

/*==============================================================================
 * MATHEMATICAL UTILITY FUNCTIONS (CUDA KERNELS)
 *============================================================================*/

__host__ __device__ void skew_matrix_4(my_real *w, my_real *S) {
    S[0] = REAL(0.0);     S[1] = -w[0]; S[2] = -w[1]; S[3] = -w[2];
    S[4] = w[0];  S[5] = REAL(0.0);     S[6] = w[2];  S[7] = -w[1];
    S[8] = w[1];  S[9] = -w[2]; S[10] = REAL(0.0);    S[11] = w[0];
    S[12] = w[2]; S[13] = w[1]; S[14] = -w[0]; S[15] = REAL(0.0);
}

__host__ __device__ void cross_product(my_real *a, my_real *b, my_real *result) {
    result[0] = a[1]*b[2] - a[2]*b[1];
    result[1] = a[2]*b[0] - a[0]*b[2]; 
    result[2] = a[0]*b[1] - a[1]*b[0];
}

__host__ __device__ my_real quaternion_norm(my_real *q) {
    return SQRT(q[0]*q[0] + q[1]*q[1] + q[2]*q[2] + q[3]*q[3]);
}

/*==============================================================================
 * DYNAMICS AND NUMERICAL INTEGRATION SCHEMES
 *============================================================================*/

__host__ __device__ void attitude_dynamics(my_real *X, my_real *U, my_real *X_dot, attitude_params *params) {
    my_real *q = X;
    my_real *w = &X[n_quat];
    
    // Quaternion kinematics: q̇ = 0.5 * S(ω) * q
    my_real S[16];
    skew_matrix_4(w, S);
    for(int i = 0; i < n_quat; i++) {
        X_dot[i] = REAL(0.5) * (S[i*4]*q[0] + S[i*4+1]*q[1] + S[i*4+2]*q[2] + S[i*4+3]*q[3]);
    }
    
    // Angular dynamics: ω̇ = I⁻¹ * (τ - ω × (I*ω))
    my_real Iw[n_vel] = {
        params->inertia[0] * w[0], 
        params->inertia[1] * w[1], 
        params->inertia[2] * w[2]
    };
    
    my_real w_cross_Iw[n_vel];
    cross_product(w, Iw, w_cross_Iw);

    for(int i = 0; i < n_vel; i++) {
        X_dot[n_quat+i] = (U[i] - w_cross_Iw[i]) / params->inertia[i];
    }
}

__host__ __device__ void rk4(my_real *X, my_real *U, my_real dt, my_real *X_next, attitude_params *params) {
    my_real k1[n_states], k2[n_states], k3[n_states], k4[n_states];
    my_real X_temp[n_states];
    
    // k1 = f(X, U)
    attitude_dynamics(X, U, k1, params);
    
    // k2 = f(X + dt/2*k1, U)
    for(int i = 0; i < n_states; i++) X_temp[i] = X[i] + dt/REAL(2.0)*k1[i];
    attitude_dynamics(X_temp, U, k2, params);

    // k3 = f(X + dt/2*k2, U)
    for(int i = 0; i < n_states; i++) X_temp[i] = X[i] + dt/REAL(2.0)*k2[i];
    attitude_dynamics(X_temp, U, k3, params);

    // k4 = f(X + dt*k3, U)
    for(int i = 0; i < n_states; i++) X_temp[i] = X[i] + dt*k3[i];
    attitude_dynamics(X_temp, U, k4, params);

    // Final integration step
    for(int i = 0; i < n_states; i++) {
        X_next[i] = X[i] + dt/REAL(6.0)*(k1[i] + REAL(2.0)*k2[i] + REAL(2.0)*k3[i] + k4[i]);
    }
    
    // Maintain quaternion unit constraint
    my_real q_norm = quaternion_norm(X_next);
    if(q_norm > REAL(1e-6)) {
        for(int i = 0; i < n_quat; i++) X_next[i] /= q_norm;
    }
}

__host__ __device__ void euler(my_real *X, my_real *U, my_real dt, my_real *X_next, attitude_params *params) {
    my_real X_dot[n_states];
    
    attitude_dynamics(X, U, X_dot, params);

    for(int i = 0; i < n_states; i++) {
        X_next[i] = X[i] + dt * X_dot[i];
    }
    
    my_real q_norm = quaternion_norm(X_next);
    if(q_norm > REAL(1e-6)) {
        for(int i = 0; i < n_quat; i++) X_next[i] /= q_norm;
    }
}

/*==============================================================================
 * FITNESS FUNCTIONS IMPLEMENTATIONS
 *==============================================================================*/

__host__ __device__ my_real fit_full(my_real *solution_vector, int particle_id, attitude_params *params) {
    my_real dt = solution_vector[PARTICLE_IDX_FULL(particle_id, DT_IDX_FULL)];
    my_real constraints_violation = REAL(0.0);
    my_real X[n_states], X_next[n_states];
    for(int i = 0; i < n_quat; i++) X[i] = params->initial_quat[i];
    for(int i = 0; i < n_vel; i++) X[n_quat+i] = params->initial_omega[i];

    int switches = 0;

    for(int step = 0; step < N_STEPS; step++) {
        my_real U[n_controls];
        for(int axis = 0; axis < n_controls; axis++) {
            int torque_idx = TORQUE_IDX(step, axis);
            U[axis] = solution_vector[PARTICLE_IDX_FULL(particle_id, torque_idx)];
            
            if (step > 0) {
                int previous_idx = TORQUE_IDX(step-1, axis);
                my_real previous_torque = solution_vector[PARTICLE_IDX_FULL(particle_id, previous_idx)];
                if (U[axis] * previous_torque < 0) {
                    switches++;
                }
            }
        }

        INTEGRATE(X, U, dt, X_next, params);

        my_real q_norm = quaternion_norm(X_next);
        constraints_violation -= QUAT_NORM_PENALTY * FABS(q_norm - REAL(1.0));

        for(int i = 0; i < n_states; i++) X[i] = X_next[i];
    }

    constraints_violation -= SWITCH_PENALTY * switches;

    my_real final_error = REAL(0.0);
    my_real diff;
    for(int i = 0; i < n_quat; i++) {
        diff = X[i] - params->target_quat[i];
        final_error += diff * diff;
    }
    for(int i = 0; i < n_vel; i++) {
        diff = X[n_quat+i] - params->target_omega[i];
        final_error += diff * diff;
    }
    final_error = SQRT(final_error);
    constraints_violation -= FINAL_STATE_PENALTY * final_error;
    my_real total_time = dt * N_STEPS;

    constraints_violation -= DT_PENALTY * total_time;
    return constraints_violation;

}

__host__ __device__ my_real fit_sto(my_real *solution_vector, int particle_id, attitude_params *params)
{
    my_real dt = solution_vector[PARTICLE_IDX_STO(particle_id, DT_IDX_STO)];
    my_real total_time = dt * N_STEPS;

    // Decode initial control signs (map [-1,1] to {-1,+1})
    my_real initial_signs[n_controls];
    for (int axis = 0; axis < n_controls; axis++)
    {
        initial_signs[axis] = solution_vector[PARTICLE_IDX_STO(particle_id, axis)];
    }

    // Decode and sort switch times for each axis (normalized to [0,1])
    my_real switch_times[n_controls][MAX_SWITCHES_PER_AXIS];
    int num_switches[n_controls];

    for (int axis = 0; axis < n_controls; axis++)
    {
        // Extract switch times for this axis
        my_real times[MAX_SWITCHES_PER_AXIS];
        for (int s = 0; s < MAX_SWITCHES_PER_AXIS; s++)
        {
            times[s] = solution_vector[PARTICLE_IDX_STO(particle_id, SWITCH_TIME_IDX(axis, s))];
            // Clamp to [0,1]
            times[s] = FMAX(REAL(0.0), FMIN(REAL(1.0), times[s]));
        }

        // Bubble sort (simple, efficient for small arrays)
        for (int i = 0; i < MAX_SWITCHES_PER_AXIS - 1; i++)
        {
            for (int j = 0; j < MAX_SWITCHES_PER_AXIS - i - 1; j++)
            {
                if (times[j] > times[j + 1])
                {
                    my_real temp = times[j];
                    times[j] = times[j + 1];
                    times[j + 1] = temp;
                }
            }
        }

        // Remove duplicates and count valid switches
        num_switches[axis] = 0;
        for (int s = 0; s < MAX_SWITCHES_PER_AXIS; s++)
        {
            // Only count if different from previous and not at boundaries
            if (times[s] > REAL(0.01) && times[s] < REAL(0.99))
            {
                if (num_switches[axis] == 0 ||
                    FABS(times[s] - switch_times[axis][num_switches[axis] - 1]) > REAL(0.01))
                {
                    switch_times[axis][num_switches[axis]] = times[s];
                    num_switches[axis]++;
                }
            }
        }
    }

    // Simulate trajectory with bang-bang control
    my_real constraints_violation = REAL(0.0);
    my_real X[n_states], X_next[n_states];

    for (int i = 0; i < n_quat; i++)
        X[i] = params->initial_quat[i];
    for (int i = 0; i < n_vel; i++)
        X[n_quat + i] = params->initial_omega[i];

    my_real current_time = REAL(0.0);

    for (int step = 0; step < N_STEPS; step++)
    {
        my_real step_start_time = current_time / total_time; // Normalized time [0,1]
        my_real U[n_controls];

        // Determine control for each axis based on switch times
        for (int axis = 0; axis < n_controls; axis++)
        {
            my_real control_sign = initial_signs[axis];

            // Count how many switches have occurred by this time
            for (int s = 0; s < num_switches[axis]; s++)
            {
                if (step_start_time >= switch_times[axis][s])
                {
                    control_sign *= REAL(-1.0); // Flip sign at each switch
                }
            }

            // Apply bang-bang control at max torque
            U[axis] = control_sign * params->max_torque;
        }

        INTEGRATE(X, U, dt, X_next, params);

        my_real q_norm = quaternion_norm(X_next);
        constraints_violation -= QUAT_NORM_PENALTY * FABS(q_norm - REAL(1.0));

        for (int i = 0; i < n_states; i++)
            X[i] = X_next[i];
        current_time += dt;
    }

    // Final state error
    my_real final_error = REAL(0.0);
    my_real diff;
    for (int i = 0; i < n_quat; i++)
    {
        diff = X[i] - params->target_quat[i];
        final_error += diff * diff;
    }
    for (int i = 0; i < n_vel; i++)
    {
        diff = X[n_quat + i] - params->target_omega[i];
        final_error += diff * diff;
    }
    final_error = SQRT(final_error);

    constraints_violation -= FINAL_STATE_PENALTY * final_error;
    constraints_violation -= DT_PENALTY * total_time;

    return constraints_violation;
}

/*==============================================================================
 * CUDA KERNEL IMPLEMENTATIONS
 *============================================================================*/

__global__ void move(my_real *position_d, my_real *velocity_d, my_real *fitness_d,
                     my_real *pbest_pos_d, my_real *pbest_fit_d,
                     particle_gbest *gbest_d, my_real *aux, my_real *aux_pos) {

    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    int tidx = threadIdx.x;
    
    extern __shared__ my_real sharedMemory[];
    my_real *privateBestQueue = (my_real *)sharedMemory;                    
    int *privateBestParticleQueue = (int *)&sharedMemory[blockDim.x];   
    __shared__ unsigned int queue_num;
    
    if (particle_id >= particle_cnt_d) return;

    if (tidx == 0) queue_num = 0;
    __syncthreads();

    hiprandState state1, state2;
    hiprand_init((unsigned long long)clock() + particle_id * 2, 0, 0, &state1);
    hiprand_init((unsigned long long)clock() + particle_id * 2 + 1, 0, 0, &state2);

    my_real w = w_d, c1 = c1_d, c2 = c2_d;
    if (decay_w_d) {
        w = w_d - (w_d - min_w_d) * (my_real)particle_id / particle_cnt_d;
    }
    if (decay_c1_d) {
        c1 = c1_d - (c1_d - min_c1_d) * (my_real)particle_id / particle_cnt_d;
    }
    if (decay_c2_d) {
        c2 = c2_d - (c2_d - min_c2_d) * (my_real)particle_id / particle_cnt_d;
    }

    // Determine index macro based on method (use dimensions_d for runtime selection)
    for (int dim = 0; dim < dimensions_d; dim++) {
        int pos_idx = particle_id * dimensions_d + dim;  // Direct calculation instead of macro
        
        my_real pos = position_d[pos_idx];
        my_real vel = velocity_d[pos_idx];
        my_real pbest_pos = pbest_pos_d[pos_idx];
        my_real gbest_pos = gbest_d->position[dim];  // Access via pointer
        
        vel = w * vel +
              c1 * CURAND(state1) * (pbest_pos - pos) +
              c2 * CURAND(state2) * (gbest_pos - pos);
        
        // Handle dimension-specific constraints
        if (dimensions_d == DIMENSIONS_FULL) {
            // FULL method
            if (dim < TORQUE_DIMS) {
                vel = FMAX(-max_v_torque_d, FMIN(max_v_torque_d, vel));
                pos = pos + vel;
                pos = FMAX(min_torque_d, FMIN(max_torque_d, pos));
            } else {  // dt dimension
                vel = FMAX(-max_v_dt_d, FMIN(max_v_dt_d, vel));
                pos = pos + vel;
                pos = FMAX(min_dt_d, FMIN(max_dt_d, pos));
            }
        } else {
            // STO method
            if (dim < N_SIGNS) {  // Sign dimensions
                my_real vel_clamp = -LOG(REAL(1.0)/saturation_d - REAL(1.0))/alpha_d;
                vel = FMAX(-vel_clamp, FMIN(vel_clamp, vel));
                my_real sigmoid = REAL(1.0) / (REAL(1.0) + EXP(-alpha_d * vel));
                pos = (CURAND(state1) < sigmoid) ? REAL(1.0) : REAL(-1.0);
            } else if (dim < N_SIGNS + N_SWITCH_TIMES) {  // Switch time dimensions
                vel = FMAX(REAL(-1.0), FMIN(REAL(1.0), vel));
                pos = pos + vel;
                pos = FMAX(REAL(0.0), FMIN(REAL(1.0), pos));
            } else {  // dt dimension
                vel = FMAX(-max_v_dt_d, FMIN(max_v_dt_d, vel));
                pos = pos + vel;
                pos = FMAX(min_dt_d, FMIN(max_dt_d, pos));
            }
        }
        
        position_d[pos_idx] = pos;
        velocity_d[pos_idx] = vel;
    }
    
    // Evaluate fitness based on method
    my_real new_fitness;
    if (dimensions_d == DIMENSIONS_FULL) {
        new_fitness = fit_full(position_d, particle_id, &att_params_d);
    } else {
        new_fitness = fit_sto(position_d, particle_id, &att_params_d);
    }
    fitness_d[particle_id] = new_fitness;

    if (new_fitness > pbest_fit_d[particle_id]) {
        pbest_fit_d[particle_id] = new_fitness;
        for (int dim = 0; dim < dimensions_d; dim++) {
            int pos_idx = particle_id * dimensions_d + dim;
            pbest_pos_d[pos_idx] = position_d[pos_idx];
        }
    }
    
    __syncthreads();

    if (new_fitness > gbest_d->fitness) {
        unsigned int my_index = atomicAdd(&queue_num, 1);
        if (my_index < blockDim.x) {
            privateBestQueue[my_index] = new_fitness;
            privateBestParticleQueue[my_index] = particle_id;
        }
    }
    
    __syncthreads();

    if (tidx == 0) {
        aux[blockIdx.x] = REAL_MIN;
        aux_pos[blockIdx.x] = -1;
        
        if (queue_num > 0) {
            my_real best_fitness = privateBestQueue[0];
            int best_idx = 0;
            
            for (unsigned int i = 1; i < queue_num && i < blockDim.x; i++) {
                if (privateBestQueue[i] > best_fitness) {
                    best_fitness = privateBestQueue[i];
                    best_idx = i;
                }
            }
            aux[blockIdx.x] = best_fitness;
            aux_pos[blockIdx.x] = privateBestParticleQueue[best_idx];
        }
    }
}

__global__ void findBest(particle_gbest *gbest, my_real *aux, my_real *aux_pos, my_real *position_d) {
    int tid = threadIdx.x;
    
    my_real my_fitness = (tid < BlocksPerGrid) ? aux[tid] : REAL_MIN;
    int my_particle = (tid < BlocksPerGrid) ? (int)aux_pos[tid] : -1;
    
    for (int offset = 16; offset > 0; offset /= 2) {
        my_real other_fitness = __shfl_down_sync(0xffffffff, my_fitness, offset);
        int other_particle = __shfl_down_sync(0xffffffff, my_particle, offset);
        if (other_fitness > my_fitness) {
            my_fitness = other_fitness;
            my_particle = other_particle;
        }
    }
    
    if (tid == 0 && my_fitness > gbest->fitness) {
        gbest->fitness = my_fitness;
        if (my_particle >= 0) {
            for (int dim = 0; dim < dimensions_d; dim++) {
                int pos_idx = my_particle * dimensions_d + dim;
                gbest->position[dim] = position_d[pos_idx];  // Access via pointer
            }
        }
        __threadfence();
    }
}

/*==============================================================================
 * PSO OPTIMIZER CLASS IMPLEMENTATION
 *============================================================================*/
PSOOptimizer::PSOOptimizer(casadi::DM& state_matrix, casadi::DM& input_matrix,
                           casadi::DM& dt_matrix, PSOMethod method, bool verbose,
                           int num_particles) 
    : configured_(false)
    , results_valid_(false)
    , max_iterations_(ITERATIONS)
    , num_particles_(num_particles)
    , inertia_weight_(W)
    , cognitive_weight_(C_1)
    , social_weight_(C_2)
    , min_w_(MIN_W)
    , min_c1_(MIN_C1)
    , min_c2_(MIN_C2)
    , decay_w_(DEC_INERTIA)
    , decay_c1_(DEC_C1)
    , decay_c2_(DEC_C2)
    , particles_(nullptr)
    , position_d_(nullptr)
    , velocity_d_(nullptr)
    , fitness_d_(nullptr)
    , pbest_pos_d_(nullptr)
    , pbest_fit_d_(nullptr)
    , gbest_d_(nullptr)
    , gbest_pos_d_(nullptr)
    , aux_(nullptr)
    , aux_pos_(nullptr)
    , lhs_samples_(nullptr)
    , lhs_generated_(false)
    , verbose_(verbose)
    , X(state_matrix)
    , U(input_matrix)
    , dt(dt_matrix)
    , method_(method)
{
    // Initialize CUDA events
    if (!handleCudaError(hipEventCreate(&start_event_), __FILE__, __LINE__) ||
        !handleCudaError(hipEventCreate(&stop_event_), __FILE__, __LINE__)) {
        std::cerr << "Failed to create CUDA events" << std::endl;
    }

    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);

    // Validate output matrix dimensions
    if ((X.size1() != n_states || X.size2() != N_STEPS + 1) ||
       (U.size1() != n_controls || U.size2() != N_STEPS) ||
       (dt.size1() != N_STEPS || dt.size2() != 1)){
        std::cerr << "Error: Output matrices have incorrect dimensions." << std::endl;
        std::cerr << "Expected dimensions | Given dimensions: " 
                  << "X: (" << n_states << ", " << N_STEPS + 1 << ") | (" << X.size1() << ", " << X.size2() << "), "
                  << "U: (" << n_controls << ", " << N_STEPS << ") | (" << U.size1() << ", " << U.size2() << "), "
                  << "dt: (" << N_STEPS << ", 1) | (" << dt.size1() << ", " << dt.size2() << ")" << std::endl;
        cleanup();
        return;
    }

    // Determine optimization dimensions based on method
    if (method_ == PSOMethod::FULL) {
        dimensions_ = DIMENSIONS_FULL; // Torques + dt
    } else if (method_ == PSOMethod::STO) {
        dimensions_ = DIMENSIONS_STO; // Initial signs + switch times + dt
    } else {
        std::cerr << "Error: Unknown PSO method." << std::endl;
        cleanup();
        return;
    }

    // Initialize attitude parameters with default spacecraft values
    memset(&att_params_, 0, sizeof(attitude_params));
    
    att_params_.inertia[0] = static_cast<my_real>(i_x);
    att_params_.inertia[1] = static_cast<my_real>(i_y);
    att_params_.inertia[2] = static_cast<my_real>(i_z);
    att_params_.max_torque = static_cast<my_real>(tau_max);
    att_params_.min_torque = -static_cast<my_real>(tau_max);
    att_params_.max_dt = static_cast<my_real>(dt_max);
    att_params_.min_dt = static_cast<my_real>(dt_min);
    
    // Initialize velocity limits
    max_v_torque_ = att_params_.max_torque - att_params_.min_torque;
    max_v_dt_ = att_params_.max_dt - att_params_.min_dt;

    // Allocate LHS samples storage
    lhs_samples_ = new my_real*[num_particles_];
    for (int i = 0; i < num_particles_; i++) {
        lhs_samples_[i] = new my_real[dimensions_];
    }

    // Allocate host particle structure
    particles_ = (particle*)malloc(sizeof(particle));
    if (!particles_) {
        std::cerr << "Failed to allocate particle structure" << std::endl;
        cleanup();
        return;
    }
    
    particles_->position = (my_real*)malloc(sizeof(my_real) * num_particles_ * dimensions_);
    particles_->velocity = (my_real*)malloc(sizeof(my_real) * num_particles_ * dimensions_);
    particles_->pbest_pos = (my_real*)malloc(sizeof(my_real) * num_particles_ * dimensions_);
    particles_->fitness = (my_real*)malloc(sizeof(my_real) * num_particles_);
    particles_->pbest_fit = (my_real*)malloc(sizeof(my_real) * num_particles_);
    
    if (!particles_->position || !particles_->velocity || !particles_->pbest_pos || 
        !particles_->fitness || !particles_->pbest_fit) {
        std::cerr << "Failed to allocate particle arrays" << std::endl;
        cleanup();
        return;
    }

    // Allocate gbest structure - NOW WITH POINTER
    gbest_ = (particle_gbest*)malloc(sizeof(particle_gbest));
    if (!gbest_) {
        std::cerr << "Failed to allocate global best particle" << std::endl;
        cleanup();
        return;
    }
    // Allocate the position array
    gbest_->position = (my_real*)malloc(sizeof(my_real) * dimensions_);
    if (!gbest_->position) {
        std::cerr << "Failed to allocate global best position array" << std::endl;
        cleanup();
        return;
    }
    gbest_->fitness = -REAL_MAX;

    // One-time GPU initialization
    if (!allocateDeviceMemory() || !copyImmutableConstants()) {
        std::cerr << "GPU initialization failed" << std::endl;
        cleanup();
        return;
    }

    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&setup_time_, start_event_, stop_event_), __FILE__, __LINE__);
    setup_time_ /= 1000.0f;
}

PSOOptimizer::~PSOOptimizer() {
    cleanup();
}

bool PSOOptimizer::allocateDeviceMemory() {
    size_t particle_data_size = sizeof(my_real) * num_particles_ * dimensions_;
    
    if (!handleCudaError(hipMalloc((void**)&position_d_, particle_data_size), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&velocity_d_, particle_data_size), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&pbest_pos_d_, particle_data_size), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&fitness_d_, sizeof(my_real) * num_particles_), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&pbest_fit_d_, sizeof(my_real) * num_particles_), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&aux_, sizeof(my_real) * BlocksPerGrid), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&aux_pos_, sizeof(my_real) * BlocksPerGrid), __FILE__, __LINE__)) {
        return false;
    }
    
    // Allocate device memory for gbest structure
    if (!handleCudaError(hipMalloc((void**)&gbest_d_, sizeof(particle_gbest)), __FILE__, __LINE__)) {
        return false;
    }
    
    // Allocate device memory for gbest position array
    if (!handleCudaError(hipMalloc((void**)&gbest_pos_d_, sizeof(my_real) * dimensions_), __FILE__, __LINE__)) {
        return false;
    }
    
    return true;
}

void PSOOptimizer::setPSOParameters(int max_iterations, double inertia_weight, 
    double cognitive_weight, double social_weight, bool decay_inertia, 
    bool decay_cognitive, bool decay_social, double min_inertia, double min_cognitive, 
    double min_social,  double sigmoid_alpha, double sigmoid_saturation) {

    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);

    max_iterations_ = max_iterations;
    inertia_weight_ = static_cast<my_real>(inertia_weight);
    cognitive_weight_ = static_cast<my_real>(cognitive_weight);
    social_weight_ = static_cast<my_real>(social_weight);
    decay_w_ = decay_inertia;   
    decay_c1_ = decay_cognitive;
    decay_c2_ = decay_social;
    min_w_ = static_cast<my_real>(min_inertia);
    min_c1_ = static_cast<my_real>(min_cognitive);
    min_c2_ = static_cast<my_real>(min_social);
    sigmoid_alpha_ = static_cast<my_real>(sigmoid_alpha);
    sigmoid_saturation_ = static_cast<my_real>(sigmoid_saturation);

    copyImmutableConstants();

    results_valid_ = false;
    // Stop timing
    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);
    
    // Calculate execution time
    float temp_time;
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&temp_time, start_event_, stop_event_), __FILE__, __LINE__);
    setup_time_ += temp_time; // Accumulate
}

bool PSOOptimizer::copyImmutableConstants() {
    // Copy PSO parameters
    if (!handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(w_d), &inertia_weight_, sizeof(my_real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(c1_d), &cognitive_weight_, sizeof(my_real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(c2_d), &social_weight_, sizeof(my_real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(decay_w_d), &decay_w_, sizeof(bool)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(decay_c1_d), &decay_c1_, sizeof(bool)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(decay_c2_d), &decay_c2_, sizeof(bool)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(min_w_d), &min_w_, sizeof(my_real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(min_c1_d), &min_c1_, sizeof(my_real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(min_c2_d), &min_c2_, sizeof(my_real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(alpha_d), &sigmoid_alpha_, sizeof(my_real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(saturation_d), &sigmoid_saturation_, sizeof(my_real)), __FILE__, __LINE__)) {
        return false;
    }
    
    // Copy physical constraints (immutable)
    if (!handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(max_torque_d), &att_params_.max_torque, sizeof(my_real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(min_torque_d), &att_params_.min_torque, sizeof(my_real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(max_dt_d), &att_params_.max_dt, sizeof(my_real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(min_dt_d), &att_params_.min_dt, sizeof(my_real)), __FILE__, __LINE__)) {
        return false;
    }
    
    // Copy velocity limits
    if (!handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(max_v_torque_d), &max_v_torque_, sizeof(my_real)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(max_v_dt_d), &max_v_dt_, sizeof(my_real)), __FILE__, __LINE__)) {
        return false;
    }
    
    // Copy dimensions
    if (!handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(particle_cnt_d), &num_particles_, sizeof(int)), __FILE__, __LINE__)) {
        return false;
    }
    
    if (!handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(dimensions_d), &dimensions_, sizeof(int)), __FILE__, __LINE__)) {
        return false;
    }
    
    return true;
}

bool PSOOptimizer::copyMutableStateParameters() {
    // Only copy the att_params structure (contains initial/target states)
    return handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(att_params_d), &att_params_, sizeof(attitude_params)), __FILE__, __LINE__);
}

void PSOOptimizer::setStates(const double* initial_state, const double* target_state) {
    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);
    // Update initial and target states
    for (int i = 0; i < n_quat; i++) {
        att_params_.initial_quat[i] = static_cast<my_real>(initial_state[i]);
        att_params_.target_quat[i] = static_cast<my_real>(target_state[i]);
    }
    for (int i = 0; i < n_vel; i++) {
        att_params_.initial_omega[i] = static_cast<my_real>(initial_state[i + n_quat]);
        att_params_.target_omega[i] = static_cast<my_real>(target_state[i + n_quat]);
    }
    
    // Copy updated parameters to device constant memory
    copyMutableStateParameters();
    
    results_valid_ = false;

    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);
    
    // Calculate execution time
    float temp_time;
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&temp_time, start_event_, stop_event_), __FILE__, __LINE__);
    setup_time_ += temp_time; // Accumulate
}

bool PSOOptimizer::initializeParticles_sto(bool regenerate_lhs) {
    srand((unsigned)time(NULL));
    
    // Generate or reuse LHS samples
    if (regenerate_lhs || !lhs_generated_) {
        generateLHSSamples(lhs_samples_);
        lhs_generated_ = true;
    }
    
    // Initialize global best
    gbest_->fitness = -REAL_MAX;
    
    int best_particle = 0;
    
    for (int i = 0; i < num_particles_; i++) {
        // Initialize sign variables
        for (int dim = 0; dim < N_SIGNS; dim++) {
            int idx = i * dimensions_ + dim;
            particles_->position[idx] = (lhs_samples_[i][dim] > REAL(0.5)) ? REAL(1.0) : REAL(-1.0);
            particles_->velocity[idx] = (lhs_samples_[i][dim] - REAL(0.5)) * REAL(4.0);
            particles_->pbest_pos[idx] = particles_->position[idx];
        }

        // Initialize switch time variables
        for (int dim = N_SIGNS; dim < N_SIGNS + N_SWITCH_TIMES; dim++) {
            int idx = i * dimensions_ + dim;
            particles_->position[idx] = lhs_samples_[i][dim];
            particles_->velocity[idx] = (lhs_samples_[i][dim] - REAL(0.5)) * REAL(2.0);
            particles_->pbest_pos[idx] = particles_->position[idx];
        }

        // Initialize dt
        int dt_idx = i * dimensions_ + (dimensions_ - 1);
        my_real dt_range = att_params_.max_dt - att_params_.min_dt;
        particles_->position[dt_idx] = lhs_samples_[i][dimensions_ - 1] * dt_range + att_params_.min_dt;
        particles_->velocity[dt_idx] = (lhs_samples_[i][dimensions_ - 1] - REAL(0.5)) * REAL(2.0) * max_v_dt_;
        particles_->pbest_pos[dt_idx] = particles_->position[dt_idx];

        // Evaluate fitness
        particles_->fitness[i] = fit_sto(particles_->position, i, &att_params_);
        particles_->pbest_fit[i] = particles_->fitness[i];

        if (i == 0 || particles_->pbest_fit[i] > gbest_->fitness) {
            best_particle = i;
            gbest_->fitness = particles_->fitness[i];
        }
    }
    
    // Copy global best position
    for (int dim = 0; dim < dimensions_; dim++) {
        int idx = best_particle * dimensions_ + dim;
        gbest_->position[dim] = particles_->position[idx];
    }
    
    return true;
}

bool PSOOptimizer::initializeParticles_full(bool regenerate_lhs) {
    srand((unsigned)time(NULL));
    
    // Generate or reuse LHS samples
    if (regenerate_lhs || !lhs_generated_) {
        generateLHSSamples(lhs_samples_);
        lhs_generated_ = true;
    }
    
    // Initialize global best
    gbest_->fitness = -REAL_MAX;
    
    int best_particle = 0;
    
    for (int i = 0; i < num_particles_; i++) {
        // Initialize torque variables
        for (int dim = 0; dim < TORQUE_DIMS; dim++) {
            int idx = i * dimensions_ + dim;
            my_real torque_range = att_params_.max_torque - att_params_.min_torque;
            
            particles_->position[idx] = lhs_samples_[i][dim] * torque_range + att_params_.min_torque;
            particles_->velocity[idx] = (lhs_samples_[i][dim] - REAL(0.5)) * REAL(2.0) * max_v_torque_;
            particles_->pbest_pos[idx] = particles_->position[idx];
        }
        
        // Initialize dt
        int dt_idx = i * dimensions_ + (dimensions_ - 1);
        my_real dt_range = att_params_.max_dt - att_params_.min_dt;
        
        particles_->position[dt_idx] = lhs_samples_[i][dimensions_ - 1] * dt_range + att_params_.min_dt;
        particles_->velocity[dt_idx] = (lhs_samples_[i][dimensions_ - 1] - REAL(0.5)) * REAL(2.0) * max_v_dt_;
        particles_->pbest_pos[dt_idx] = particles_->position[dt_idx];
        
        // Evaluate fitness
        particles_->fitness[i] = fit_full(particles_->position, i, &att_params_);
        particles_->pbest_fit[i] = particles_->fitness[i];
        
        if (i == 0 || particles_->pbest_fit[i] > gbest_->fitness) {
            best_particle = i;
            gbest_->fitness = particles_->fitness[i];
        }
    }
    
    
    // Copy global best position
    for (int dim = 0; dim < dimensions_; dim++) {
        int idx = best_particle * dimensions_ + dim;
        gbest_->position[dim] = particles_->position[idx];
    }
    
    return true;
}

void PSOOptimizer::generateLHSSamples(my_real** samples) {
    // Generate LHS samples in [0,1] for each dimension
    for (int dim = 0; dim < dimensions_; dim++) {
        std::vector<my_real> intervals(num_particles_);
        
        // Create stratified intervals
        for (int i = 0; i < num_particles_; i++) {
            my_real interval_start = static_cast<my_real>(i) / num_particles_;
            my_real interval_width = REAL(1.0) / num_particles_;
            intervals[i] = interval_start + RND() * interval_width;
        }
        
        // Shuffle to break correlations
        for (int i = num_particles_ - 1; i > 0; i--) {
            int j = rand() % (i + 1);
            std::swap(intervals[i], intervals[j]);
        }
        
        // Assign to samples matrix
        for (int i = 0; i < num_particles_; i++) {
            samples[i][dim] = intervals[i];
        }
    }
}

bool PSOOptimizer::optimize(bool regenerate_lhs) {

    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);
    
    // Initialize particles
    if (method_ == PSOMethod::FULL) {
        if (!initializeParticles_full(regenerate_lhs)) {
            std::cerr << "Particle initialization failed" << std::endl;
            return false;
        }
    } else  if (method_ == PSOMethod::STO) {
        if (!initializeParticles_sto(regenerate_lhs)) {
            std::cerr << "Particle initialization failed" << std::endl;
            return false;
        }
    }
    
    // Copy gbest position pointer to device
    // First copy the position array, then update the struct pointer
    if (!handleCudaError(hipMemcpy(gbest_pos_d_, gbest_->position, sizeof(my_real) * dimensions_, hipMemcpyHostToDevice), __FILE__, __LINE__)) {
        return false;
    }
    
    // Create temporary host struct with device pointer
    particle_gbest temp_gbest;
    temp_gbest.position = gbest_pos_d_;  // Point to device memory
    temp_gbest.fitness = gbest_->fitness;
    
    // Copy struct to device
    if (!handleCudaError(hipMemcpy(gbest_d_, &temp_gbest, sizeof(particle_gbest), hipMemcpyHostToDevice), __FILE__, __LINE__)) {
        return false;
    }
    
    // Copy initial data to device
    size_t particle_data_size = sizeof(my_real) * num_particles_ * dimensions_;
    if (!handleCudaError(hipMemcpy(position_d_, particles_->position, particle_data_size, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(velocity_d_, particles_->velocity, particle_data_size, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(pbest_pos_d_, particles_->pbest_pos, particle_data_size, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(fitness_d_, particles_->fitness, sizeof(my_real) * num_particles_, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(pbest_fit_d_, particles_->pbest_fit, sizeof(my_real) * num_particles_, hipMemcpyHostToDevice), __FILE__, __LINE__)) {
        return false;
    }
    
    if (verbose_) {
        std::cout << "Starting PSO optimization..." << std::endl;
        std::cout << "LHS: " << (regenerate_lhs ? "Regenerated" : "Reused") << std::endl;
        std::cout << "Initial best fitness: " << gbest_->fitness << std::endl;
    }
    
    // Main optimization loop
    int shared_mem_size = sizeof(my_real) * ThreadsPerBlock + sizeof(int) * ThreadsPerBlock;

    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);
    float temp_time;
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&temp_time, start_event_, stop_event_), __FILE__, __LINE__);
    setup_time_ += temp_time / 1000.0f;  // Convert ms to seconds
    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);

    for (int iter = 0; iter < max_iterations_; iter++) {
        move<<<BlocksPerGrid, ThreadsPerBlock, shared_mem_size>>>(
            position_d_, velocity_d_, fitness_d_, pbest_pos_d_, pbest_fit_d_,
            gbest_d_, aux_, aux_pos_);
        
        if (!handleCudaError(hipDeviceSynchronize(), __FILE__, __LINE__)) {
            return false;
        }
        
        findBest<<<1, 32>>>(gbest_d_, aux_, aux_pos_, position_d_);
        
        if (!handleCudaError(hipDeviceSynchronize(), __FILE__, __LINE__)) {
            return false;
        }
        
        if (verbose_ && (iter % 100 == 0 || iter == max_iterations_ - 1)) {
            particle_gbest current_best;
            if (handleCudaError(hipMemcpy(&current_best, gbest_d_, sizeof(particle_gbest), hipMemcpyDeviceToHost), __FILE__, __LINE__)) {
                std::cout << "Iteration " << iter << ": Best fitness = " << current_best.fitness << std::endl;
            }
        }
    }
    
    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&exec_time_, start_event_, stop_event_), __FILE__, __LINE__);
    
    // Copy final results
    if (!handleCudaError(hipMemcpy(gbest_->position, gbest_pos_d_, sizeof(my_real) * dimensions_, hipMemcpyDeviceToHost), __FILE__, __LINE__)) {
        std::cerr << "Error copying final gbest position" << std::endl;
        return false;
    }
    
    // Copy fitness value
    if (!handleCudaError(hipMemcpy(&gbest_->fitness, &gbest_d_->fitness, sizeof(my_real), hipMemcpyDeviceToHost), __FILE__, __LINE__)) {
        std::cerr << "Error copying final gbest fitness" << std::endl;
        return false;
    }

    dt_opt_ = gbest_->position[dimensions_ - 1];  // Last dimension is dt
    total_time_ = dt_opt_ * N_STEPS;
    final_fitness_ = gbest_->fitness;
    exec_time_ = exec_time_ / 1000.0f;

    if (method_ == PSOMethod::STO) {
        if (!extractResults_sto()) {
            std::cerr << "Error extracting STO results" << std::endl;
            return false;
        }
    } else if (method_ == PSOMethod::FULL) {
        if (!extractResults_full()) {
            std::cerr << "Error extracting FULL results" << std::endl;
            return false;
        }
    }

    results_valid_ = true;
    configured_ = true;

    if (verbose_) {
        printResults();
    }

    return true;
}

bool PSOOptimizer::extractResults_sto() {
    double dt_double = static_cast<double>(dt_opt_);
    my_real current_state[n_states], next_state[n_states];
    
    // Initialize with initial conditions
    for (int i = 0; i < n_quat; i++) {
        current_state[i] = att_params_.initial_quat[i];
        X(i, 0) = static_cast<double>(current_state[i]);
    }
    for (int i = 0; i < n_vel; i++) {
        current_state[n_quat + i] = att_params_.initial_omega[i];
        X(n_quat + i, 0) = static_cast<double>(current_state[n_quat + i]);
    }
    // Extract STO bang-bang trajectory
    my_real initial_signs[n_controls];
    for (int axis = 0; axis < n_controls; axis++) {
        initial_signs[axis] = gbest_->position[axis];
    }

    my_real switch_times[n_controls][MAX_SWITCHES_PER_AXIS];
    int num_switches[n_controls];

    for (int axis = 0; axis < n_controls; axis++) {
        my_real times[MAX_SWITCHES_PER_AXIS];
        for (int s = 0; s < MAX_SWITCHES_PER_AXIS; s++) {
            times[s] = gbest_->position[N_SIGNS + axis * MAX_SWITCHES_PER_AXIS + s];
            times[s] = std::max(REAL(0.0), std::min(REAL(1.0), times[s]));
        }

        // Sort times
        for (int i = 0; i < MAX_SWITCHES_PER_AXIS - 1; i++) {
            for (int j = 0; j < MAX_SWITCHES_PER_AXIS - i - 1; j++) {
                if (times[j] > times[j + 1]) {
                    my_real temp = times[j];
                    times[j] = times[j + 1];
                    times[j + 1] = temp;
                }
            }
        }

        num_switches[axis] = 0;
        for (int s = 0; s < MAX_SWITCHES_PER_AXIS; s++) {
            if (times[s] > REAL(0.01) && times[s] < REAL(0.99)) {
                if (num_switches[axis] == 0 ||
                    std::abs(times[s] - switch_times[axis][num_switches[axis] - 1]) > REAL(0.01)) {
                    switch_times[axis][num_switches[axis]] = times[s];
                    num_switches[axis]++;
                }
            }
        }
    }

    my_real total_time = dt_opt_ * N_STEPS;
    my_real current_time = REAL(0.0);

    for (int step = 0; step < N_STEPS; step++) {
        my_real step_start_time = current_time / total_time;
        my_real controls[n_controls];

        for (int axis = 0; axis < n_controls; axis++) {
            my_real control_sign = initial_signs[axis];

            for (int s = 0; s < num_switches[axis]; s++) {
                if (step_start_time >= switch_times[axis][s]) {
                    control_sign *= REAL(-1.0);
                }
            }

            controls[axis] = control_sign * att_params_.max_torque;
            U(axis, step) = static_cast<double>(controls[axis]);
        }

        INTEGRATE(current_state, controls, dt_opt_, next_state, &att_params_);

        for (int i = 0; i < n_states; i++) {
            current_state[i] = next_state[i];
            X(i, step + 1) = static_cast<double>(current_state[i]);
        }

        dt(step) = dt_double;
        current_time += dt_opt_;
    }

    // Validate final state
    my_real final_error = REAL(0.0);
    my_real diff;
    for(int i = 0; i < n_quat; i++) {
        diff = current_state[i] - att_params_.target_quat[i];
        final_error += diff * diff;
    }
    for(int i = 0; i < n_vel; i++) {
        diff = current_state[n_quat+i] - att_params_.target_omega[i];
        final_error += diff * diff;
    }
    final_error = SQRT(final_error);
    
    if (final_error > REAL(1e-3) || verbose_) {
        std::cerr << "Warning: Final state deviates significantly from target state. Final error: " 
                  << final_error << std::endl;
    }
    
    return true;
}

bool PSOOptimizer::extractResults_full() {
    double dt_double = static_cast<double>(dt_opt_);
    my_real current_state[n_states], next_state[n_states];
    
    // Initialize with initial conditions
    for (int i = 0; i < n_quat; i++) {
        current_state[i] = att_params_.initial_quat[i];
        X(i, 0) = static_cast<double>(current_state[i]);
    }
    for (int i = 0; i < n_vel; i++) {
        current_state[n_quat + i] = att_params_.initial_omega[i];
        X(n_quat + i, 0) = static_cast<double>(current_state[n_quat + i]);
    }

    // Extract FULL control trajectory
    for (int step = 0; step < N_STEPS; step++) {
        my_real controls[n_controls];
        for (int axis = 0; axis < n_controls; axis++) {
            controls[axis] = gbest_->position[step * n_controls + axis];
            U(axis, step) = static_cast<double>(controls[axis]);
        }

        INTEGRATE(current_state, controls, dt_opt_, next_state, &att_params_);

        for (int i = 0; i < n_states; i++) {
            current_state[i] = next_state[i];
            X(i, step + 1) = static_cast<double>(current_state[i]);
        }

        dt(step) = dt_double;
    }
    
    // Validate final state
    my_real final_error = REAL(0.0);
    my_real diff;
    for(int i = 0; i < n_quat; i++) {
        diff = current_state[i] - att_params_.target_quat[i];
        final_error += diff * diff;
    }
    for(int i = 0; i < n_vel; i++) {
        diff = current_state[n_quat+i] - att_params_.target_omega[i];
        final_error += diff * diff;
    }
    final_error = SQRT(final_error);
    
    if (final_error > REAL(1e-3) || verbose_) {
        std::cerr << "Warning: Final state deviates significantly from target state. Final error: " 
                  << final_error << std::endl;
    }
    
    return true;
}

void PSOOptimizer::cleanup() {
    // Free LHS samples
    if (lhs_samples_) {
        for (int i = 0; i < num_particles_; i++) {
            if (lhs_samples_[i]) delete[] lhs_samples_[i];
        }
        delete[] lhs_samples_;
        lhs_samples_ = nullptr;
    }
    
    // Free host memory
    if (particles_) {
        if (particles_->position) free(particles_->position);
        if (particles_->velocity) free(particles_->velocity);
        if (particles_->pbest_pos) free(particles_->pbest_pos);
        if (particles_->fitness) free(particles_->fitness);
        if (particles_->pbest_fit) free(particles_->pbest_fit);
        free(particles_);
        particles_ = nullptr;
    }
    
    // Free gbest - FREE THE POSITION ARRAY FIRST
    if (gbest_) {
        if (gbest_->position) {
            free(gbest_->position);
            gbest_->position = nullptr;
        }
        free(gbest_);
        gbest_ = nullptr;
    }
    
    // Free device memory
    if (position_d_) { hipFree(position_d_); position_d_ = nullptr; }
    if (velocity_d_) { hipFree(velocity_d_); velocity_d_ = nullptr; }
    if (fitness_d_) { hipFree(fitness_d_); fitness_d_ = nullptr; }
    if (pbest_pos_d_) { hipFree(pbest_pos_d_); pbest_pos_d_ = nullptr; }
    if (pbest_fit_d_) { hipFree(pbest_fit_d_); pbest_fit_d_ = nullptr; }
    if (gbest_pos_d_) { hipFree(gbest_pos_d_); gbest_pos_d_ = nullptr; }  // Free position array
    if (gbest_d_) { hipFree(gbest_d_); gbest_d_ = nullptr; }  // Free struct
    if (aux_) { hipFree(aux_); aux_ = nullptr; }
    if (aux_pos_) { hipFree(aux_pos_); aux_pos_ = nullptr; }

    if (start_event_) { hipEventDestroy(start_event_); start_event_ = nullptr; }
    if (stop_event_) { hipEventDestroy(stop_event_); stop_event_ = nullptr; }
}

bool PSOOptimizer::handleCudaError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err)
                  << " at " << file << ":" << line << std::endl;
        return false;
    }
    return true;
}

void PSOOptimizer::printResults() const
{
    if (!results_valid_)
    {
        std::cout << "No valid results available." << std::endl;
        return;
    }

    std::cout << "\n=== PSO Optimization Results ===" << std::endl;
    std::cout << "Final fitness: " << std::setprecision(6) << final_fitness_ << std::endl;
    std::cout << "Total maneuver time: " << total_time_ << " seconds" << std::endl;
    std::cout << "Time step: " << dt_opt_ << " seconds" << std::endl;
    std::cout << "Execution time: " << exec_time_ << " seconds" << std::endl;
    std::cout << "Setup time: " << setup_time_ << " seconds" << std::endl;
    std::cout << "Total computation time: " << (setup_time_ + exec_time_) << " seconds" << std::endl;
    std::cout << "===============================\n"
              << std::endl;
}