#include "hip/hip_runtime.h"
/**
 * @file pso_optimizer.cu
 * @brief C++ Class Implementation for CUDA-accelerated PSO Spacecraft Attitude Control
 * 
 * This implementation provides the complete class functionality for PSO optimization
 * of spacecraft attitude maneuvers with clean input/output interfaces.
 * 
 * @author Leonardo Eitner
 * @date 11/09/2025
 * @version 2.0
 */

/*==============================================================================
 * INCLUDES
 *============================================================================*/
#include "test2.h"
using ::erfinv;
/*==============================================================================
 * CUDA CONSTANT MEMORY DECLARATIONS
 *============================================================================*/

/** @brief PSO algorithm parameters in device constant memory */
__constant__ float w_d, c1_d, c2_d;

/** @brief Physical constraint bounds in device constant memory */
__constant__ float max_torque_d, min_torque_d;
__constant__ float max_dt_d, min_dt_d;

/** @brief PSO velocity limits in device constant memory */
__constant__ float max_v_torque_d, max_v_dt_d;

/** @brief Problem dimensions in device constant memory */
__constant__ int particle_cnt_d, dimensions_d;

/** @brief Complete attitude parameters structure in device constant memory */
__constant__ attitude_params att_params_d;

/*==============================================================================
 * MATHEMATICAL UTILITY FUNCTIONS (CUDA KERNELS)
 *============================================================================*/

__host__ __device__ void skew_matrix_4(float *w, float *S) {
    S[0] = 0;     S[1] = -w[0]; S[2] = -w[1]; S[3] = -w[2];
    S[4] = w[0];  S[5] = 0;     S[6] = w[2];  S[7] = -w[1];
    S[8] = w[1];  S[9] = -w[2]; S[10] = 0;    S[11] = w[0];
    S[12] = w[2]; S[13] = w[1]; S[14] = -w[0]; S[15] = 0;
}

__host__ __device__ void cross_product(float *a, float *b, float *result) {
    result[0] = a[1]*b[2] - a[2]*b[1];
    result[1] = a[2]*b[0] - a[0]*b[2]; 
    result[2] = a[0]*b[1] - a[1]*b[0];
}

__host__ __device__ float quaternion_norm(float *q) {
    return sqrt(q[0]*q[0] + q[1]*q[1] + q[2]*q[2] + q[3]*q[3]);
}

__host__ __device__ void attitude_dynamics(float *X, float *U, float *X_dot, attitude_params *params) {
    float *q = X;
    float *w = &X[n_quat];
    
    // Quaternion kinematics: q̇ = 0.5 * S(ω) * q
    float S[16];
    skew_matrix_4(w, S);
    for(int i = 0; i < n_quat; i++) {
        X_dot[i] = 0.5f * (S[i*4]*q[0] + S[i*4+1]*q[1] + S[i*4+2]*q[2] + S[i*4+3]*q[3]);
    }
    
    // Angular dynamics: ω̇ = I⁻¹ * (τ - ω × (I*ω))
    float Iw[n_vel] = {
        params->inertia[0] * w[0], 
        params->inertia[1] * w[1], 
        params->inertia[2] * w[2]
    };
    
    float w_cross_Iw[n_vel];
    cross_product(w, Iw, w_cross_Iw);

    for(int i = 0; i < n_vel; i++) {
        X_dot[n_quat+i] = (U[i] - w_cross_Iw[i]) / params->inertia[i];
    }
}

__host__ __device__ void rk4(float *X, float *U, float dt, float *X_next, attitude_params *params) {
    float k1[n_states], k2[n_states], k3[n_states], k4[n_states];
    float X_temp[n_states];
    
    attitude_dynamics(X, U, k1, params);
    
    for(int i = 0; i < n_states; i++) X_temp[i] = X[i] + dt/2.0f*k1[i];
    attitude_dynamics(X_temp, U, k2, params);

    for(int i = 0; i < n_states; i++) X_temp[i] = X[i] + dt/2.0f*k2[i];
    attitude_dynamics(X_temp, U, k3, params);

    for(int i = 0; i < n_states; i++) X_temp[i] = X[i] + dt*k3[i];
    attitude_dynamics(X_temp, U, k4, params);

    for(int i = 0; i < n_states; i++) {
        X_next[i] = X[i] + dt/6.0f*(k1[i] + 2.0f*k2[i] + 2.0f*k3[i] + k4[i]);
    }
    
    float q_norm = quaternion_norm(X_next);
    if(q_norm > 1e-6f) {
        for(int i = 0; i < n_quat; i++) X_next[i] /= q_norm;
    }
}

__host__ __device__ void euler(float *X, float *U, float dt, float *X_next, attitude_params *params) {
    float X_dot[n_states];
    
    attitude_dynamics(X, U, X_dot, params);

    for(int i = 0; i < n_states; i++) {
        X_next[i] = X[i] + dt * X_dot[i];
    }
    
    float q_norm = quaternion_norm(X_next);
    if(q_norm > 1e-6f) {
        for(int i = 0; i < n_quat; i++) X_next[i] /= q_norm;
    }
}

__host__ __device__ float fit(float *solution_vector, int particle_id, attitude_params *params) {
    float dt = solution_vector[PARTICLE_POS_IDX(particle_id, DT_IDX)];
    
    float X[n_states], X_next[n_states];
    for(int i = 0; i < n_quat; i++) X[i] = params->initial_quat[i];
    for(int i = 0; i < n_vel; i++) X[n_quat+i] = params->initial_omega[i];

    float constraint_violation = 0.0f;
    int switches = 0;

    for(int step = 0; step < N_STEPS; step++) {
        float U[n_controls];
        for(int axis = 0; axis < n_controls; axis++) {
            int torque_idx = TORQUE_IDX(step, axis);
            U[axis] = solution_vector[PARTICLE_POS_IDX(particle_id, torque_idx)];
            
            if (step > 0) {
                int previous_idx = TORQUE_IDX(step-1, axis);
                float previous_torque = solution_vector[PARTICLE_POS_IDX(particle_id, previous_idx)];
                if (U[axis] * previous_torque < 0) {
                    switches++;
                }
            }
        }
        
        euler(X, U, dt, X_next, params);

        float q_norm = quaternion_norm(X_next);
        constraint_violation += QUAT_NORM_PENALTY * fabsf(q_norm - 1.0f);

        for(int i = 0; i < n_states; i++) X[i] = X_next[i];
    }

    constraint_violation += SWITCH_PENALTY * switches;
    
    float final_error = 0.0f;
    for(int i = 0; i < n_quat; i++) {
        final_error += pow(X[i] - params->target_quat[i], 2);
    }
    for(int i = 0; i < n_vel; i++) {
        final_error += pow(X[n_quat+i] - params->target_omega[i], 2);
    }
    final_error = sqrt(final_error);
    constraint_violation += FINAL_STATE_PENALTY * final_error;
    printf("Particle %d final error: %f\n", particle_id, final_error);
    float total_time = dt * N_STEPS;
    
    return -DT_PENALTY * total_time - constraint_violation;
}

/*==============================================================================
 * CUDA KERNEL IMPLEMENTATIONS
 *============================================================================*/

__global__ void move(float *position_d, float *velocity_d, float *fitness_d,
                     float *pbest_pos_d, float *pbest_fit_d, 
                     particle_gbest *gbest_d, float *aux, float *aux_pos) {
    
    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    int tidx = threadIdx.x;
    
    extern __shared__ float sharedMemory[];
    float *privateBestQueue = (float *)sharedMemory;                    
    int *privateBestParticleQueue = (int *)&sharedMemory[blockDim.x];   
    __shared__ unsigned int queue_num;
    
    if (particle_id >= particle_cnt_d) return;

    if (tidx == 0) queue_num = 0;
    __syncthreads();

    hiprandState state1, state2;
    hiprand_init((unsigned long long)clock() + particle_id * 2, 0, 0, &state1);
    hiprand_init((unsigned long long)clock() + particle_id * 2 + 1, 0, 0, &state2);

    float w = w_d;
    if (DEC_INERTIA) {
        w = w_d - (w_d - MIN_W) * particle_id / N_PARTICLES;
    }
    
    for (int dim = 0; dim < dimensions_d; dim++) {
        int pos_idx = PARTICLE_POS_IDX(particle_id, dim);
        
        float pos = position_d[pos_idx];
        float vel = velocity_d[pos_idx];
        float pbest_pos = pbest_pos_d[pos_idx];
        float gbest_pos = gbest_d->position[dim];
        
        vel = w * vel +
              c1_d * hiprand_uniform(&state1) * (pbest_pos - pos) +
              c2_d * hiprand_uniform(&state2) * (gbest_pos - pos);
        
        if (dim < TORQUE_DIMS) {
            vel = fmax(-max_v_torque_d, fmin(max_v_torque_d, vel));
            pos = pos + vel;
            pos = fmax(min_torque_d, fmin(max_torque_d, pos));
        } else {
            vel = fmax(-max_v_dt_d, fmin(max_v_dt_d, vel));
            pos = pos + vel;
            pos = fmax(min_dt_d, fmin(max_dt_d, pos));
        }
        
        position_d[pos_idx] = pos;
        velocity_d[pos_idx] = vel;
    }
    
    float new_fitness = fit(position_d, particle_id, &att_params_d);
    fitness_d[particle_id] = new_fitness;
    
    if (new_fitness > pbest_fit_d[particle_id]) {
        pbest_fit_d[particle_id] = new_fitness;
        for (int dim = 0; dim < dimensions_d; dim++) {
            pbest_pos_d[PARTICLE_POS_IDX(particle_id, dim)] = 
                position_d[PARTICLE_POS_IDX(particle_id, dim)];
        }
    }
    
    __syncthreads();

    if (new_fitness > gbest_d->fitness) {
        unsigned int my_index = atomicAdd(&queue_num, 1);
        if (my_index < blockDim.x) {
            privateBestQueue[my_index] = new_fitness;
            privateBestParticleQueue[my_index] = particle_id;
        }
    }
    
    __syncthreads();

    if (tidx == 0) {
        aux[blockIdx.x] = -FLT_MAX;
        aux_pos[blockIdx.x] = -1;
        
        if (queue_num > 0) {
            float best_fitness = privateBestQueue[0];
            int best_idx = 0;
            
            for (unsigned int i = 1; i < queue_num && i < blockDim.x; i++) {
                if (privateBestQueue[i] > best_fitness) {
                    best_fitness = privateBestQueue[i];
                    best_idx = i;
                }
            }
            
            aux[blockIdx.x] = best_fitness;
            aux_pos[blockIdx.x] = privateBestParticleQueue[best_idx];
        }
    }
}

__global__ void findBest(particle_gbest *gbest, float *aux, float *aux_pos, float *position_d) {
    int tid = threadIdx.x;
    
    float my_fitness = (tid < BlocksPerGrid) ? aux[tid] : -FLT_MAX;
    int my_particle = (tid < BlocksPerGrid) ? (int)aux_pos[tid] : -1;
    
    for (int offset = 16; offset > 0; offset /= 2) {
        float other_fitness = __shfl_down_sync(0xffffffff, my_fitness, offset);
        int other_particle = __shfl_down_sync(0xffffffff, my_particle, offset);
        if (other_fitness > my_fitness) {
            my_fitness = other_fitness;
            my_particle = other_particle;
        }
    }
    
    if (tid == 0 && my_fitness > gbest->fitness) {
        gbest->fitness = my_fitness;
        if (my_particle >= 0) {
            for (int dim = 0; dim < DIMENSIONS; dim++) {
                gbest->position[dim] = position_d[PARTICLE_POS_IDX(my_particle, dim)];
            }
        }
        __threadfence();
    }
}

/*==============================================================================
 * PSO OPTIMIZER CLASS IMPLEMENTATION
 *============================================================================*/

PSOOptimizer::PSOOptimizer(const double* initial_state, const double* target_state, bool verbose) 
    : configured_(false)
    , results_valid_(false)
    , max_iterations_(MAX_ITERA)
    , num_particles_(N_PARTICLES)
    , inertia_weight_(W)
    , cognitive_weight_(C1)
    , social_weight_(C2)
    , particles_(nullptr)
    , position_d_(nullptr)
    , velocity_d_(nullptr)
    , fitness_d_(nullptr)
    , pbest_pos_d_(nullptr)
    , pbest_fit_d_(nullptr)
    , gbest_d_(nullptr)
    , aux_(nullptr)
    , aux_pos_(nullptr)
    , verbose_(verbose)
{
    // Initialize CUDA events
    if (!handleCudaError(hipEventCreate(&start_event_), __FILE__, __LINE__) ||
        !handleCudaError(hipEventCreate(&stop_event_), __FILE__, __LINE__)) {
        std::cerr << "Failed to create CUDA events" << std::endl;
    }

    // Start timing
    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);

    // Initialize attitude parameters with default values
    memset(&att_params_, 0, sizeof(attitude_params));
    
    // Set default spacecraft parameters (will be overridden by setSpacecraftParameters)
    att_params_.inertia[0] = static_cast<float>(i_x);
    att_params_.inertia[1] = static_cast<float>(i_y);
    att_params_.inertia[2] = static_cast<float>(i_z);
    att_params_.max_torque = static_cast<float>(tau_max);
    att_params_.min_torque = -static_cast<float>(tau_max);
    att_params_.max_dt = static_cast<float>(dt_max);
    att_params_.min_dt = static_cast<float>(dt_min);
    for (int i = 0; i < n_quat; i++) {
        att_params_.initial_quat[i] = static_cast<float>(initial_state[i]);
        att_params_.target_quat[i] = static_cast<float>(target_state[i]);
        printf("Initial quat[%d]: %f, Target quat[%d]: %f\n", i, att_params_.initial_quat[i], i, att_params_.target_quat[i]);
    }
    for (int i = 0; i < n_vel; i++) {
        att_params_.initial_omega[i] = static_cast<float>(initial_state[i + n_quat]);
        att_params_.target_omega[i] = static_cast<float>(target_state[i + n_quat]);
        printf("Initial omega[%d]: %f, Target omega[%d]: %f\n", i, att_params_.initial_omega[i], i, att_params_.target_omega[i]);
    }
    
    // Initialize velocity limits
    max_v_torque_ = 2.0f * att_params_.max_torque;
    max_v_dt_ = att_params_.max_dt - att_params_.min_dt;

    // Validate configuration
    if (!validateConfiguration()) {
        std::cerr << "Configuration validation failed" << std::endl;
        cleanup();
    }

        // Stop timing
    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);
    
    // Calculate execution time
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&setup_time_, start_event_, stop_event_), __FILE__, __LINE__);
}

PSOOptimizer::~PSOOptimizer() {
    cleanup();
}

void PSOOptimizer::setPSOParameters(int max_iterations, int num_particles,
                                   double inertia_weight, double cognitive_weight, double social_weight) {
    
    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);

    num_particles_ = num_particles;
    inertia_weight_ = static_cast<float>(inertia_weight);
    cognitive_weight_ = static_cast<float>(cognitive_weight);
    social_weight_ = static_cast<float>(social_weight);

    results_valid_ = false;
    // Stop timing
    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);
    
    // Calculate execution time
    float temp_time;
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&temp_time, start_event_, stop_event_), __FILE__, __LINE__);
    setup_time_ += temp_time; // Accumulate
}

bool PSOOptimizer::validateConfiguration() const {
    // Check that all required parameters are set
    bool valid = true;
    
    // Check inertia values
    if (att_params_.inertia[0] <= 0 || att_params_.inertia[1] <= 0 || att_params_.inertia[2] <= 0) {
        std::cerr << "Error: Inertia values must be positive" << std::endl;
        valid = false;
    }
    
    // Check torque limits
    if (att_params_.max_torque <= 0) {
        std::cerr << "Error: Maximum torque must be positive" << std::endl;
        valid = false;
    }
    
    // Check time constraints
    if (att_params_.min_dt < 0 || att_params_.max_dt <= att_params_.min_dt) {
        std::cerr << "Error: Invalid time step constraints" << std::endl;
        valid = false;
    }
    
    // Check quaternion normalization
    float q_norm_initial = sqrt(att_params_.initial_quat[0]*att_params_.initial_quat[0] + 
                               att_params_.initial_quat[1]*att_params_.initial_quat[1] + 
                               att_params_.initial_quat[2]*att_params_.initial_quat[2] + 
                               att_params_.initial_quat[3]*att_params_.initial_quat[3]);
    
    float q_norm_target = sqrt(att_params_.target_quat[0]*att_params_.target_quat[0] + 
                              att_params_.target_quat[1]*att_params_.target_quat[1] + 
                              att_params_.target_quat[2]*att_params_.target_quat[2] + 
                              att_params_.target_quat[3]*att_params_.target_quat[3]);
    
    if (fabs(q_norm_initial - 1.0f) > 1e-3f) {
        std::cerr << "Warning: Initial quaternion is not normalized (norm = " << q_norm_initial << ")" << std::endl;
    }
    
    if (fabs(q_norm_target - 1.0f) > 1e-3f) {
        std::cerr << "Warning: Target quaternion is not normalized (norm = " << q_norm_target << ")" << std::endl;
    }
    
    return valid;
}

bool PSOOptimizer::initializeCUDA() {
    // Allocate device memory
    size_t particle_data_size = sizeof(float) * num_particles_ * DIMENSIONS;
    
    if (!handleCudaError(hipMalloc((void**)&position_d_, particle_data_size), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&velocity_d_, particle_data_size), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&pbest_pos_d_, particle_data_size), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&fitness_d_, sizeof(float) * num_particles_), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&pbest_fit_d_, sizeof(float) * num_particles_), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&gbest_d_, sizeof(particle_gbest)), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&aux_, sizeof(float) * BlocksPerGrid), __FILE__, __LINE__) ||
        !handleCudaError(hipMalloc((void**)&aux_pos_, sizeof(float) * BlocksPerGrid), __FILE__, __LINE__)) {
        return false;
    }
    
    // Copy constants to device constant memory
    if (!handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(w_d), &inertia_weight_, sizeof(float)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(c1_d), &cognitive_weight_, sizeof(float)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(c2_d), &social_weight_, sizeof(float)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(max_torque_d), &att_params_.max_torque, sizeof(float)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(min_torque_d), &att_params_.min_torque, sizeof(float)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(max_dt_d), &att_params_.max_dt, sizeof(float)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(min_dt_d), &att_params_.min_dt, sizeof(float)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(max_v_torque_d), &max_v_torque_, sizeof(float)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(max_v_dt_d), &max_v_dt_, sizeof(float)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(particle_cnt_d), &num_particles_, sizeof(int)), __FILE__, __LINE__)) {
        return false;
    }
    
    int dimensions = DIMENSIONS;
    if (!handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(dimensions_d), &dimensions, sizeof(int)), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(att_params_d), &att_params_, sizeof(attitude_params)), __FILE__, __LINE__)) {
        return false;
    }
    
    return true;
}

bool PSOOptimizer::initializeParticles() {
    srand((unsigned)time(NULL));
    
    // Allocate host memory
    particles_ = (particle*)malloc(sizeof(particle));
    if (!particles_) {
        std::cerr << "Failed to allocate particle structure" << std::endl;
        return false;
    }
    
    particles_->position = (float*)malloc(sizeof(float) * num_particles_ * DIMENSIONS);
    particles_->velocity = (float*)malloc(sizeof(float) * num_particles_ * DIMENSIONS);
    particles_->pbest_pos = (float*)malloc(sizeof(float) * num_particles_ * DIMENSIONS);
    particles_->fitness = (float*)malloc(sizeof(float) * num_particles_);
    particles_->pbest_fit = (float*)malloc(sizeof(float) * num_particles_);
    
    if (!particles_->position || !particles_->velocity || !particles_->pbest_pos || 
        !particles_->fitness || !particles_->pbest_fit) {
        std::cerr << "Failed to allocate particle arrays" << std::endl;
        return false;
    }
    
    // Initialize global best
    gbest_.fitness = -FLT_MAX;
    
    // Initialize each particle
    int best_particle = 0;
    for (int i = 0; i < num_particles_; i++) {
        // Initialize torque variables
        for (int dim = 0; dim < TORQUE_DIMS; dim++) {
            int idx = PARTICLE_POS_IDX(i, dim);
            float torque_range = att_params_.max_torque - att_params_.min_torque;
            
            particles_->position[idx] = RND() * torque_range + att_params_.min_torque;
            particles_->velocity[idx] = (RND() - 0.5f) * 2.0f * max_v_torque_;
            particles_->pbest_pos[idx] = particles_->position[idx];
        }
        
        // Initialize time step variable
        int dt_idx = PARTICLE_POS_IDX(i, DT_IDX);
        float dt_range = att_params_.max_dt - att_params_.min_dt;
        
        particles_->position[dt_idx] = RND() * dt_range + att_params_.min_dt;
        particles_->velocity[dt_idx] = (RND() - 0.5f) * 2.0f * max_v_dt_;
        particles_->pbest_pos[dt_idx] = particles_->position[dt_idx];
        
        // Evaluate initial fitness
        particles_->fitness[i] = fit(particles_->position, i, &att_params_);
        particles_->pbest_fit[i] = particles_->fitness[i];
        
        // Track global best
        if (i == 0 || particles_->pbest_fit[i] > gbest_.fitness) {
            best_particle = i;
            gbest_.fitness = particles_->fitness[i];
        }
    }
    
    // Copy global best position
    for (int dim = 0; dim < DIMENSIONS; dim++) {
        int idx = PARTICLE_POS_IDX(best_particle, dim);
        gbest_.position[dim] = particles_->position[idx];
    }
    
    return true;
}

bool PSOOptimizer::optimize(double* X, double* U, double* dt) {

    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);
    
    // Initialize CUDA and particles
    if (!initializeCUDA() || !initializeParticles()) {
        std::cerr << "Initialization failed" << std::endl;
        cleanup();
        return false;
    }
    
    // Copy initial data to device
    size_t particle_data_size = sizeof(float) * num_particles_ * DIMENSIONS;
    if (!handleCudaError(hipMemcpy(position_d_, particles_->position, particle_data_size, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(velocity_d_, particles_->velocity, particle_data_size, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(pbest_pos_d_, particles_->pbest_pos, particle_data_size, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(fitness_d_, particles_->fitness, sizeof(float) * num_particles_, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(pbest_fit_d_, particles_->pbest_fit, sizeof(float) * num_particles_, hipMemcpyHostToDevice), __FILE__, __LINE__) ||
        !handleCudaError(hipMemcpy(gbest_d_, &gbest_, sizeof(particle_gbest), hipMemcpyHostToDevice), __FILE__, __LINE__)) {
        cleanup();
        return false;
    }
    
    if (verbose_) {
        std::cout << "Starting PSO optimization..." << std::endl;
        std::cout << "Particles: " << num_particles_ << ", Dimensions: " << DIMENSIONS << std::endl;
        std::cout << "Max iterations: " << max_iterations_ << std::endl;
        std::cout << "Initial best fitness: " << gbest_.fitness << std::endl;
    }
    
    // Main optimization loop
    int shared_mem_size = sizeof(float) * ThreadsPerBlock + sizeof(int) * ThreadsPerBlock;

    // Stop timing
    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);
    
    // Calculate execution time
    float temp_time;
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&temp_time, start_event_, stop_event_), __FILE__, __LINE__);
    setup_time_ += temp_time; // Accumulate

    // Start timing
    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);

    for (int iter = 0; iter < max_iterations_; iter++) {
        // Launch PSO update kernel
        move<<<BlocksPerGrid, ThreadsPerBlock, shared_mem_size>>>(
            position_d_, velocity_d_, fitness_d_, pbest_pos_d_, pbest_fit_d_,
            gbest_d_, aux_, aux_pos_);
        
        if (!handleCudaError(hipDeviceSynchronize(), __FILE__, __LINE__)) {
            cleanup();
            return false;
        }
        
        // Launch global best finding kernel
        findBest<<<1, 32>>>(gbest_d_, aux_, aux_pos_, position_d_);
        
        if (!handleCudaError(hipDeviceSynchronize(), __FILE__, __LINE__)) {
            cleanup();
            return false;
        }
        
        // Progress reporting
        if (verbose_ && (iter % 100 == 0 || iter == max_iterations_ - 1)) {
            particle_gbest current_best;
            if (handleCudaError(hipMemcpy(&current_best, gbest_d_, sizeof(particle_gbest), hipMemcpyDeviceToHost), __FILE__, __LINE__)) {
                std::cout << "Iteration " << iter << ": Best fitness = " << current_best.fitness << std::endl;
            }
        }
    }
    
    // Stop timing
    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);

    // Calculate execution time
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&exec_time_, start_event_, stop_event_), __FILE__, __LINE__);
    
    handleCudaError(hipEventRecord(start_event_), __FILE__, __LINE__);

    // Copy final results
    if (!handleCudaError(hipMemcpy(&gbest_, gbest_d_, sizeof(particle_gbest), hipMemcpyDeviceToHost), __FILE__, __LINE__)) {
        cleanup();
        return false;
    }

    extractResults(X, U, dt);

    dt_opt_ = gbest_.position[DT_IDX];
    total_time_ = dt_opt_ * N_STEPS;
    final_fitness_ = gbest_.fitness;
    exec_time_ = exec_time_ / 1000.0f; // Convert to seconds

    results_valid_ = true;
    configured_ = true;
    
    if (verbose_) {
        std::cout << "Optimization completed!" << std::endl;
        std::cout << "Final fitness: " << final_fitness_ << std::endl;
        std::cout << "Execution time: " << exec_time_ << " seconds" << std::endl;
    }
    
    // Stop timing
    handleCudaError(hipEventRecord(stop_event_), __FILE__, __LINE__);
    
    // Calculate execution time
    handleCudaError(hipEventSynchronize(stop_event_), __FILE__, __LINE__);
    handleCudaError(hipEventElapsedTime(&temp_time, start_event_, stop_event_), __FILE__, __LINE__);
    setup_time_ += temp_time; // Accumulate
    setup_time_ /= 1000.0f; // Convert to seconds

    return true;
}

void PSOOptimizer::extractResults(double* X, double* U, double* dt) {

    double dt_double = static_cast<double>(dt_opt_);

    // Simulate trajectory using RK4 for high accuracy
    float current_state[n_states], next_state[n_states];
    
    // Initialize with initial conditions
    for (int i = 0; i < n_quat; i++) {
        current_state[i] = att_params_.initial_quat[i];
        X[i] = static_cast<double>(current_state[i]);
    }
    for (int i = 0; i < n_vel; i++) {
        current_state[n_quat + i] = att_params_.initial_omega[i];
        X[n_quat + i] = static_cast<double>(current_state[n_quat + i]);
    }

    for (int step = 0; step < N_STEPS; step++) {
        float controls[n_controls];
        int torque_idx = TORQUE_IDX(step, 0);
        for (int axis = 0; axis < n_controls; axis++) {
            controls[axis] = gbest_.position[torque_idx + axis];
        }
        
        euler(current_state, controls, dt_opt_, next_state, &att_params_);
        
        // Store next state
        for (int i = 0; i < n_states; i++) {
            current_state[i] = next_state[i];
            X[n_states * (step + 1) + i] = static_cast<double>(current_state[i]);
        }

        for (int axis = 0; axis < n_controls; axis++) {
            U[torque_idx + axis] = static_cast<double>(gbest_.position[torque_idx + axis]);
        }

        dt[step] = dt_double;
    }
}

bool PSOOptimizer::getStats(double& final_fitness, double& setup_time, double& exec_time) const {
    if (!results_valid_) {
        std::cerr << "Warning: No valid results available. Call optimize() first." << std::endl;
        return false;
    }

    final_fitness = static_cast<double>(final_fitness_);
    setup_time = static_cast<double>(setup_time_);
    exec_time = static_cast<double>(exec_time_);

    return true;
}

void PSOOptimizer::printResults() const {
    if (!results_valid_) {
        std::cout << "No valid results available." << std::endl;
        return;
    }
    
    std::cout << "\n=== PSO Optimization Results ===" << std::endl;
    std::cout << "Final fitness: " << std::setprecision(6) << final_fitness_ << std::endl;
    std::cout << "Total maneuver time: " << total_time_ << " seconds" << std::endl;
    std::cout << "Time step: " << dt_opt_ << " seconds" << std::endl;
    std::cout << "Execution time: " << exec_time_ << " seconds" << std::endl;
    std::cout << "Setup time: " << setup_time_ << " seconds" << std::endl;
    std::cout << "Total computation time: " << (setup_time_ + exec_time_) << " seconds" << std::endl;
    std::cout << "===============================\n" << std::endl;

}

void PSOOptimizer::reset() {
    cleanup();
    results_valid_ = false;
    configured_ = false;
}

void PSOOptimizer::cleanup() {
    // Free host memory
    if (particles_) {
        if (particles_->position) free(particles_->position);
        if (particles_->velocity) free(particles_->velocity);
        if (particles_->pbest_pos) free(particles_->pbest_pos);
        if (particles_->fitness) free(particles_->fitness);
        if (particles_->pbest_fit) free(particles_->pbest_fit);
        free(particles_);
        particles_ = nullptr;
    }
    
    // Free device memory
    if (position_d_) { hipFree(position_d_); position_d_ = nullptr; }
    if (velocity_d_) { hipFree(velocity_d_); velocity_d_ = nullptr; }
    if (fitness_d_) { hipFree(fitness_d_); fitness_d_ = nullptr; }
    if (pbest_pos_d_) { hipFree(pbest_pos_d_); pbest_pos_d_ = nullptr; }
    if (pbest_fit_d_) { hipFree(pbest_fit_d_); pbest_fit_d_ = nullptr; }
    if (gbest_d_) { hipFree(gbest_d_); gbest_d_ = nullptr; }
    if (aux_) { hipFree(aux_); aux_ = nullptr; }
    if (aux_pos_) { hipFree(aux_pos_); aux_pos_ = nullptr; }

    // Destroy CUDA events
    if (start_event_) { hipEventDestroy(start_event_); start_event_ = nullptr; }
    if (stop_event_) { hipEventDestroy(stop_event_); stop_event_ = nullptr; }
}

bool PSOOptimizer::handleCudaError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) 
                  << " in " << file << " at line " << line << std::endl;
        return false;
    }
    return true;
}

int main(int argc, char* argv[]) {
    int result = mainish(argc, argv);
    return result;
}